#include "hip/hip_runtime.h"
/*
 * RunSimulation.cpp
 *
 *  Created on: Jun 27, 2012
 *      Author: mwalker
 */

#include <stdio.h>
#include <math.h>
#include <time.h>
#include "DeviceStructs.h"
#include <hip/hip_runtime_api.h>

//Constant memory variables

__device__ __constant__ double PARAMS_DEVICE[N_PARAMETERS_REAL];

//CUDA function declarations

//Sets all elements in array b to zero
__global__ void resetVectorToZero(double* b, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx<N) b[idx] = 0;
}

//Element-wise addition of b*alpha to a
__global__ void addMultVectors(double* a, double* b, double alpha, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx<N) a[idx] += alpha*b[idx];
}

__global__ void ResetResiduals(hiprandState * randStates, double* R) {

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	hiprandState localState = randStates[idx];
	R[idx] = log((double)hiprand_uniform( &localState ));
	randStates[idx] =localState;

}

//Fast buffering
__global__ void UpdateFastBuffer(int N_Ele, double* grid_ca, double* buffer_ca, double* TTSurfaceArea,double* V0)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx<N_Ele) {
	  double btothi = TTSurfaceArea[idx]*PARAMS_DEVICE[I_B_TOT_SL_HI] / (V0[idx]*(1e-15));
	  double btotlo= TTSurfaceArea[idx]*PARAMS_DEVICE[I_B_TOT_SL_LO] / (V0[idx]*(1e-15));
	  double B = 1 + (btothi*PARAMS_DEVICE[I_K_D_SL_HI]/((PARAMS_DEVICE[I_K_D_SL_HI]+grid_ca[idx])*(PARAMS_DEVICE[I_K_D_SL_HI]+grid_ca[idx]))) +
			  	     (btotlo*PARAMS_DEVICE[I_K_D_SL_LO]/((PARAMS_DEVICE[I_K_D_SL_LO]+grid_ca[idx])*(PARAMS_DEVICE[I_K_D_SL_LO]+grid_ca[idx])));
	  buffer_ca[idx] /= B;
	/*if (TTSurfaceArea[idx] > 0) { //electrodiffusion approximation (DO NOT USE)
		buffer_ca[idx] /= 38.95;
	}*/
  }
}

//Reaction terms
__global__ void UpdateReactionTerms(int N_Ele, double* grid,
									double* buffer,
									double* GV0,
									double* aij,
									int* nij,
									int* TropC, double* V0, double* TTSurfaceArea,
									double* SRSurfaceArea, double* Boundary_Rates,
									int* bJSR, double Vol_JSR) {

	int j = blockIdx.x*blockDim.x + threadIdx.x;

	if (j < N_Ele) {
		double c_ca = grid[INDEX_CA*N_Ele + j];
		int b_idx;
		double r_net, c_b;

		int idx_ca = INDEX_CA*N_Ele;
		int idx_atp = INDEX_ATP*N_Ele;
		int idx_cmdn = INDEX_CMDN*N_Ele;
		int idx_trpn = INDEX_TRPN*N_Ele;
		int idx_csqn = INDEX_CSQN*N_Ele;
		int idx_dye = INDEX_DYE*N_Ele;

		//Reset buffer values for immobile buffers that may not be set
		buffer[idx_trpn + j] = 0;
		buffer[idx_csqn + j] = 0;

		//Update diffusion
		double a0, a1, a2, a3, a4; //Coefficients
		a0 = aij[5*j];
		a1 = aij[5*j+1];
		a2 = aij[5*j+2];
		a3 = aij[5*j+3];
		a4 = aij[5*j+4];
		int n1, n2, n3, n4; //Neighbors
		n1 = nij[4*j];
		n2 = nij[4*j+1];
		n3 = nij[4*j+2];
		n4 = nij[4*j+3];

		int n0 = idx_ca + j;
		buffer[n0] = PARAMS_DEVICE[I_D_CA]*(c_ca*a0 + grid[idx_ca + n1]*a1 + grid[idx_ca + n2]*a2 + grid[idx_ca + n3]*a3 + grid[idx_ca + n4]*a4) + Boundary_Rates[n0];
		/*buffer[n0] += grid[idx_ca + n1]*a1*PARAMS_DEVICE[I_D_CA];
		buffer[n0] += grid[idx_ca + n2]*a2*PARAMS_DEVICE[I_D_CA];
		buffer[n0] += grid[idx_ca + n3]*a3*PARAMS_DEVICE[I_D_CA];
		buffer[n0] += grid[idx_ca + n4]*a4*PARAMS_DEVICE[I_D_CA];
		buffer[n0] += Boundary_Rates[n0];*/

		n0 = idx_dye + j;
		//buffer[n0] = grid[n0]*a0*PARAMS_DEVICE[I_D_DYE];
		buffer[n0] = PARAMS_DEVICE[I_D_DYE]*(grid[n0]*a0+ grid[idx_dye + n1]*a1+ grid[idx_dye + n2]*a2+ grid[idx_dye + n3]*a3+ grid[idx_dye + n4]*a4)+ Boundary_Rates[n0];
		/*buffer[n0] += grid[idx_dye + n1]*a1*PARAMS_DEVICE[I_D_DYE];
		buffer[n0] += grid[idx_dye + n2]*a2*PARAMS_DEVICE[I_D_DYE];
		buffer[n0] += grid[idx_dye + n3]*a3*PARAMS_DEVICE[I_D_DYE];
		buffer[n0] += grid[idx_dye + n4]*a4*PARAMS_DEVICE[I_D_DYE];
		buffer[n0] += Boundary_Rates[n0];*/

		n0 = idx_atp + j;
		//buffer[n0] = grid[n0]*a0*PARAMS_DEVICE[I_D_ATP];
		buffer[n0] = PARAMS_DEVICE[I_D_ATP]*(grid[n0]*a0+ grid[idx_atp + n1]*a1+ grid[idx_atp + n2]*a2+ grid[idx_atp + n3]*a3+ grid[idx_atp + n4]*a4)+ Boundary_Rates[n0];
		/*buffer[n0] += grid[idx_atp + n1]*a1*PARAMS_DEVICE[I_D_ATP];
		buffer[n0] += grid[idx_atp + n2]*a2*PARAMS_DEVICE[I_D_ATP];
		buffer[n0] += grid[idx_atp + n3]*a3*PARAMS_DEVICE[I_D_ATP];
		buffer[n0] += grid[idx_atp + n4]*a4*PARAMS_DEVICE[I_D_ATP];
		buffer[n0] += Boundary_Rates[n0];*/

		n0 = idx_cmdn + j;
		//buffer[n0] = grid[n0]*a0*PARAMS_DEVICE[I_D_CMDN];
		buffer[n0] = PARAMS_DEVICE[I_D_CMDN]*(grid[n0]*a0+ grid[idx_cmdn + n1]*a1+ grid[idx_cmdn + n2]*a2+ grid[idx_cmdn + n3]*a3+ grid[idx_cmdn + n4]*a4)+ Boundary_Rates[n0];
		/*buffer[n0] += grid[idx_cmdn + n1]*a1*PARAMS_DEVICE[I_D_CMDN];
		buffer[n0] += grid[idx_cmdn + n2]*a2*PARAMS_DEVICE[I_D_CMDN];
		buffer[n0] += grid[idx_cmdn + n3]*a3*PARAMS_DEVICE[I_D_CMDN];
		buffer[n0] += grid[idx_cmdn + n4]*a4*PARAMS_DEVICE[I_D_CMDN];
		buffer[n0] += Boundary_Rates[n0];*/

		if (!bJSR[j]) {
			//Troponin C, SR, and SERCA
			if (TropC[j]) {

				b_idx = idx_trpn + j;
				c_b = grid[b_idx];
				r_net = PARAMS_DEVICE[I_K_ON_TRPN]*(PARAMS_DEVICE[I_B_TOT_TRPN]-c_b)*c_ca - PARAMS_DEVICE[I_K_OFF_TRPN]*c_b;
				buffer[idx_ca + j] += -r_net;
				buffer[idx_trpn + j] = r_net;

				//SERCA pump in same zone as troponin C
				double K_i = c_ca / PARAMS_DEVICE[I_K_D_i];
				K_i = K_i*K_i;
				double K_sr = GV0[GLOBAL_CA_NSR] / PARAMS_DEVICE[I_K_D_SR];
				K_sr = K_sr*K_sr;
				double D_cycle = 0.104217 + 17.923*K_sr + K_i*(1.75583e6 + K_sr*7.61673e6) + K_i*K_i*(6.08463e11 + K_sr*4.50544e11);
				double v_cycle = (K_i*K_i*3.24873e12 + K_i*(9.17846e6 - 11478.2*K_sr) - 0.329904*K_sr) / D_cycle;
				double d_ca = (2e-3) * v_cycle * PARAMS_DEVICE[I_A_P]; //uM / ms (Note it is given in uM/s in Williams et al 2011)
				buffer[idx_ca + j] -= d_ca;

			}
			//Dye buffering
			b_idx = idx_dye + j;
			c_b = grid[b_idx];
			r_net = PARAMS_DEVICE[I_K_ON_DYE]*(PARAMS_DEVICE[I_B_TOT_DYE]-c_b)*c_ca - PARAMS_DEVICE[I_K_OFF_DYE]*c_b;
			buffer[idx_ca + j] += -r_net;
			buffer[idx_dye + j] += r_net;

			//ATP buffering
			b_idx = idx_atp + j;
			c_b = grid[b_idx];
			r_net = PARAMS_DEVICE[I_K_ON_ATP]*(PARAMS_DEVICE[I_B_TOT_ATP]-c_b)*c_ca - PARAMS_DEVICE[I_K_OFF_ATP]*c_b;
			buffer[idx_ca + j] += -r_net;
			buffer[idx_atp + j] += r_net;

			//Calmodulin buffering
			b_idx = idx_cmdn + j;
			c_b = grid[b_idx];
			r_net = PARAMS_DEVICE[I_K_ON_CMDN]*(PARAMS_DEVICE[I_B_TOT_CMDN]-c_b)*c_ca - PARAMS_DEVICE[I_K_OFF_CMDN]*c_b;
			buffer[idx_ca + j] += -r_net;
			buffer[idx_cmdn + j] += r_net;

		} else {

			//Calsequestrin buffering
			b_idx = idx_csqn + j;
			c_b = grid[b_idx];
			r_net = PARAMS_DEVICE[I_K_ON_CSQN]*(PARAMS_DEVICE[I_B_T_JSR]-c_b)*c_ca - PARAMS_DEVICE[I_K_OFF_CSQN]*c_b;
			buffer[idx_ca + j] += -r_net;
			buffer[idx_csqn + j] = r_net;

			//JSR refill
			//buffer_ca[j] += (GV0[GLOBAL_CA_NSR] - c_ca) * PARAMS_DEVICE[I_V_REFILL] * PARAMS_DEVICE[I_V_CELL] / Vol_JSR;
			buffer[idx_ca + j] += (GV0[GLOBAL_CA_NSR] - c_ca) * PARAMS_DEVICE[I_V_REFILL]; //Volume-independent refill

		}


	}

}

__global__ void UpdateRyRFlux(double* Grid_Ca, double *Grid_Buffer,
							   double* GV0, double* V0,
							   int N_RyR, int* RyR_States, int* RyR_Ele,
							   int* RyR_JSR_Ele) {

	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < N_RyR) {

		//RyR flux
		if (RyR_States[i]) {
			double JRyR = PARAMS_DEVICE[I_V_RYR] * (Grid_Ca[RyR_JSR_Ele[i]] - Grid_Ca[RyR_Ele[i]])* PARAMS_DEVICE[I_V_CELL];
			//double JRyR = 5.1822 / V_DYAD; //1pA constant flux
			//double JRyR = (0.2*1e-6*1e15)/(1e3*2.0*96485.3365);

			Grid_Buffer[RyR_Ele[i]] += JRyR / V0[RyR_Ele[i]];
			Grid_Buffer[RyR_JSR_Ele[i]] -= JRyR / V0[RyR_JSR_Ele[i]];
		}

	}
}

__global__ void UpdateLCCFlux(double* Grid_Ca, double *Grid_Buffer,
							   double* GV0, double* V0,
							   int N_LCC, int* LCC_States, int* LCCV_States, int* LCC_Ele) {

	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < N_LCC && (LCC_States[i] == STATE_LCC_OPEN_1 || LCC_States[i] == STATE_LCC_OPEN_2) && LCCV_States[i] == STATE_LCCV_OPEN) {

		const double P_CaL = (9.13e-13)*(1e12)*(1e-3); //um^3 / ms
		const double F = 96.5; // C / mmol
		const double RT = 310 * 8.314; // J / mol
		const double F_over_RT = F/RT;
		double V = GV0[GLOBAL_VM];

		//LCC Flux
		double CA_DYAD = Grid_Ca[LCC_Ele[i]];

		if (fabs(V) < 1e-6) {
			V = 1e-6;
		}
		double J_lcc = - P_CaL * V * (2*F_over_RT) * (CA_DYAD*exp(2*V*F_over_RT) - 0.34*PARAMS_DEVICE[I_CA_0]) / (exp(2*V*F_over_RT)-1) / V0[LCC_Ele[i]];
		Grid_Buffer[LCC_Ele[i]] += J_lcc;

	}
}

__global__ void UpdateRyRStates(double dt, int N_RyR, int* RyR_States,
								double* Grid_Ca, int* RyR_Ele,
								hiprandState* randStates, double* R,
								double* gv0, int* RyR_Neighb,
								int* RyR_JSR_Ele) {

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < N_RyR) {
		int N_Open = 0;
		int N_Closed = 0;
		for (int i = 0; i < 4; i++) {
			if (RyR_Neighb[idx*4 + i] >= 0) {
				if (RyR_States[RyR_Neighb[idx*4 + i]]) {
					N_Open++;
				} else {
					N_Closed++;
				}
			}
		}

		double X, rate;
		double ca_ss = Grid_Ca[RyR_Ele[idx]];

		if (RyR_States[idx]) {
			//Open -> Closed rate
			X = exp( 0.5*PARAMS_DEVICE[I_RYR_A_STAR]*(N_Open*PARAMS_DEVICE[I_RYR_EPS_OO] - N_Closed*PARAMS_DEVICE[I_RYR_EPS_CC]) );
			rate = X*PARAMS_DEVICE[I_RYR_K_MINUS];

		} else {
			//Closed -> Open rate
			//double phi = PARAMS_DEVICE[I_RYR_PHI_M]*Grid_Ca[RyR_JSR_Ele[idx]] + PARAMS_DEVICE[I_RYR_PHI_B];
			//double phi = (pow(PARAMS_DEVICE[I_RYR_PHI_B],PARAMS_DEVICE[I_RYR_PHI_M])+pow(Grid_Ca[RyR_JSR_Ele[idx]],PARAMS_DEVICE[I_RYR_PHI_M]))/(pow(PARAMS_DEVICE[I_RYR_PHI_B],PARAMS_DEVICE[I_RYR_PHI_M])+pow(1000.0,PARAMS_DEVICE[I_RYR_PHI_M]));
			double phi = (1.0 - pow(1000.0/PARAMS_DEVICE[I_RYR_PHI_B],PARAMS_DEVICE[I_RYR_PHI_M])) + pow(Grid_Ca[RyR_JSR_Ele[idx]]/PARAMS_DEVICE[I_RYR_PHI_B],PARAMS_DEVICE[I_RYR_PHI_M]);
			X = exp( 0.5*PARAMS_DEVICE[I_RYR_A_STAR]*(N_Closed*PARAMS_DEVICE[I_RYR_EPS_CC] - N_Open*PARAMS_DEVICE[I_RYR_EPS_OO]) );
			rate = X*phi*PARAMS_DEVICE[I_RYR_K_PLUS]* pow(ca_ss,PARAMS_DEVICE[I_RYR_ETA]);
			//double p_open = Grid_Ca[RyR_JSR_Ele[idx]] / (Grid_Ca[RyR_JSR_Ele[idx]] + 650);
			//rate *= p_open;
			//rate = min(rate,PARAMS_DEVICE[I_RYR_K_MINUS]/10);

		}

		R[idx] += rate*dt;

		if (R[idx] > 0) {
			//Generate new residual
			hiprandState localState = randStates[idx];
			R[idx] = log((double)hiprand_uniform( &localState ));
			randStates[idx] = localState;

			if (RyR_States[idx] == 1) {
				RyR_States[idx] = 0;
			} else {
				RyR_States[idx] = 1;
/*
				localState = randStates[idx];
				double runif = (double)hiprand_uniform( &localState );
				randStates[idx] = localState;
				double p_open = Grid_Ca[RyR_JSR_Ele[idx]] / (Grid_Ca[RyR_JSR_Ele[idx]] + 650);
				if (runif < p_open) {
				}*/

			}
		}

	}

}

__global__ void UpdateLCCStates(double dt, int N_LCC, int N_RyR,
								int* LCC_States, int* LCCV_States,
								int* LCC_Ele, double* Grid_Ca,
								hiprandState* randStates, double* R,
								double* gv0) {

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < N_LCC) {


		double V = gv0[GLOBAL_VM];

		double ca_ss = Grid_Ca[LCC_Ele[idx]];

		const double fL=0.85; // transition	rate into open state (1/ms)
		const double gL=2.0; //	transition rate	out	of open	state (1/ms)
//		const double gPhosph = 0.049;
		const double fLprime=0.005;	// transition rate into	Ca mode	open state (1/ms)
		const double gLprime=7.0; // transition	rate out of	Ca mode	open state (1/ms)
		const double bL=1.9356;	// mode	transition parameter
		const double bL2=bL*bL;
		const double bL3=bL*bL*bL;
		const double bL4=bL*bL*bL*bL;
		const double aL=2.0; //	mode transition	parameter
		const double aL2=aL*aL;
		const double aL3=aL*aL*aL;
		const double aL4=aL*aL*aL*aL;
		const double omega=0.83*2.0*1.3*0.01;  // mode transition parameter	(1/ms)

		const double alphacf=4.0*1.2*0.416;
		const double betacf=4.0*0.45*0.049;
		const double gammacf=0.83*1.9*1.3*0.31*7.5*0.09233 / 1000.0;	// (ms-1 uM-1)

		const double CCa0_to_C0	= omega;		// = omega
		const double CCa1_to_C1	= omega/bL;	// = omega/bL
		const double CCa2_to_C2	= omega/bL2;	// = omega/bL^2
		const double CCa3_to_C3	= omega/bL3;	// = omega/bL^3
		const double CCa4_to_C4	= omega/bL4;	// = omega/bL^4

		const double yCa_frac=0.4;	// asymptotic value	for	fraction of	LCCs that

		double alpha =	alphacf	* exp(0.012*(V-35.0));
		double beta = betacf *	exp(-0.05*(V-35.0));
		double alpha_prime	= aL*alpha;
		double beta_prime = beta/bL;

		double gamma_rate =	gammacf*ca_ss;

		int i_lcc = idx + N_RyR;
		int lcc_state = LCC_States[idx];

		int target_states[3];
		double trans_rates[3];

		//LCC state
		if (lcc_state == 1) {
			trans_rates[0] = 4.0*alpha;
			trans_rates[1] = gamma_rate;
			trans_rates[2] = 0;
			target_states[0] = 2;
			target_states[1] = 7;
			target_states[2] = 0;
		} else if (lcc_state == 2) {
			trans_rates[0] = beta;
			trans_rates[1] = 3.0*alpha;
			trans_rates[2] = aL*gamma_rate;
			target_states[0] = 1;
			target_states[1] = 3;
			target_states[2] = 8;
		} else if (lcc_state == 3) {
			trans_rates[0] = 2.0*beta;
			trans_rates[1] = 2.0*alpha;
			trans_rates[2] = aL2*gamma_rate;
			target_states[0] = 2;
			target_states[1] = 4;
			target_states[2] = 9;
		} else if (lcc_state == 4) {
			trans_rates[0] = 3.0*beta;
			trans_rates[1] = alpha;
			trans_rates[2] = aL3*gamma_rate;
			target_states[0] = 3;
			target_states[1] = 5;
			target_states[2] = 10;
		} else if (lcc_state == 5) {
			trans_rates[0] = 4.0*beta;
			trans_rates[1] = fL;
			trans_rates[2] = aL4*gamma_rate;
			target_states[0] = 4;
			target_states[1] = 6;
			target_states[2] = 11;
		} else if (lcc_state == 6) {
			trans_rates[0] = gL;
			trans_rates[1] = 0;
			trans_rates[2] = 0;
			target_states[0] = 5;
			target_states[1] = 0;
			target_states[2] = 0;
		} else if (lcc_state == 7) {
			trans_rates[0] = CCa0_to_C0;
			trans_rates[1] = 4.0*alpha_prime;
			trans_rates[2] = 0;
			target_states[0] = 1;
			target_states[1] = 8;
			target_states[2] = 0;
		} else if (lcc_state == 8) {
			trans_rates[0] = beta_prime;
			trans_rates[1] = CCa1_to_C1;
			trans_rates[2] = 3.0*alpha_prime;
			target_states[0] = 7;
			target_states[1] = 2;
			target_states[2] = 9;
		} else if (lcc_state == 9) {
			trans_rates[0] = 2.0*beta_prime;
			trans_rates[1] = CCa2_to_C2;
			trans_rates[2] = 2.0*alpha_prime;
			target_states[0] = 8;
			target_states[1] = 3;
			target_states[2] = 10;
		} else if (lcc_state == 10) {
			trans_rates[0] = 3.0*beta_prime;
			trans_rates[1] = CCa3_to_C3;
			trans_rates[2] = alpha_prime;
			target_states[0] = 9;
			target_states[1] = 4;
			target_states[2] = 11;
		} else if (lcc_state == 11) {
			trans_rates[0] = 4.0*beta_prime;
			trans_rates[1] = CCa4_to_C4;
			trans_rates[2] = fLprime;
			target_states[0] = 10;
			target_states[1] = 5;
			target_states[2] = 12;
		} else if (lcc_state == 12) {
			trans_rates[0] = gLprime;
			trans_rates[1] = 0;
			trans_rates[2] = 0;
			target_states[0] = 11;
			target_states[1] = 0;
			target_states[2] = 0;
		}

		//LCC V-dependent inactivation gate
		double lccv_rate;
		int lccv_state = LCCV_States[idx];
	    double yCa_inf	= yCa_frac/(1.0+exp((V + 12.5)/5.0)) + (1.0-yCa_frac);
	    double tau_yCa	= 60.0 + 340.0/(1.0	+ exp((V+30.0)/12.0));
		if (lccv_state == 1) { //Inactivated state
			lccv_rate = yCa_inf/tau_yCa;
		} else { //Activated state
			lccv_rate = (1.0-yCa_inf)/tau_yCa;
		}

		R[i_lcc] += (trans_rates[0] + trans_rates[1] + trans_rates[2] + lccv_rate)*dt;

		if (R[i_lcc] > 0) {

			//Generate new residual
			hiprandState localState = randStates[i_lcc];
			R[i_lcc] = log((double)hiprand_uniform( &localState ));
			randStates[i_lcc] = localState;

			//Generate random uniform
			localState = randStates[i_lcc];
			double runif = (double)hiprand_uniform(&localState) * (trans_rates[0] + trans_rates[1] + trans_rates[2] + lccv_rate);
			randStates[i_lcc] = localState;

			//Update state
			if ( runif < trans_rates[0]) {
				LCC_States[idx] = target_states[0];
			} else if ( runif < trans_rates[0]+trans_rates[1]) {
				LCC_States[idx] = target_states[1];
			} else if ( runif < trans_rates[0] + trans_rates[1] + trans_rates[2]) {
				LCC_States[idx] = target_states[2];
			} else {
				LCCV_States[idx] = (lccv_state == 1) ? 2 : 1;
			}
		}

	}
}

void RunSimulation(SimData* sd, ParamStruct* ps) {

	fprintf(stdout,"Initializing simulation...\n");
	
	//Enumerate GPU devices
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, sd->device);

	printf("Using device %d with compute capability %d.%d.\n", sd->device, deviceProp.major, deviceProp.minor);
	hipSetDevice(sd->device);

	//Declare device data structures
	DeviceDomainStruct dds;
	DeviceGlobalStruct dgs;

	if ( hipSuccess != hipGetLastError() )
		fprintf(stderr,"*****************Cuda Error setting up device!\n");
	//Initialize data structures
	fprintf(stdout,"Initializing cytosol domain (%d)...\n",sd->device);
	InitializeDomainStruct(&dds, &sd->Grid, ps->Integers[I_FLAG_SECONDARY_GRADIENT], ps);

	if ( hipSuccess != hipGetLastError() )
		fprintf(stderr,"*****************Cuda Error in InitializeDomainStruct()!\n");

	fprintf(stdout,"Initializing global domain (%d)...\n",sd->device);
	InitializeDeviceGlobalStruct(&dgs, sd, ps);

	if ( hipSuccess != hipGetLastError() )
		fprintf(stderr,"*****************Cuda Error in InitializeDeviceGlobalStruct()!\n");

	//Initialize device constants
	fprintf(stdout,"Initializing device constants (%d)...\n",sd->device);
	InitializeDeviceConstants(ps);

	if ( hipSuccess != hipGetLastError() )
		fprintf(stderr,"*****************Cuda Error in InitializeDeviceConstants!\n");

	//Initialize output files
	CreateOutputFiles(sd->ensemble_index,sd->device,ps);

	//Simulation variables
	double t = 0; //time (ms)
	int output_counter = 0; //index for grid outputs

	//Time step size
	double dt = ps->Reals[I_T_STEP];
	double dt_last = 0;

	fprintf(stdout,"Starting simulation (%d)...\n",sd->device);

	//Output initial state
	printf("Writing grid files (t = %f, #%d).\n",t,output_counter);
	WriteToGridFiles(output_counter, t, &sd->Grid, sd, ps);
	output_counter++;
	printf("Writing state files (t = %f).\n",t);
	WriteToStateFiles(t, sd, ps);

	//Timer
	clock_t clock_start = clock();

	//Keep track of number of open RyRs, terminate when none are open
	int N_RyR_Open = 0;
	for (int i = 0; i < sd->N_RyR; i++) {
		N_RyR_Open += sd->RyR_States[i];
	}

	//Start CUDA profiler
	hipProfilerStart();

	//Pull in some parameters for readability
	int OUTPUT_GRID_FLAG = ps->Integers[I_OUTPUT_GRID_FLAG];
	int FLAG_FIDELITY_SIM = ps->Integers[I_FLAG_FIDELITY_SIM];
	double T_FINAL = ps->Reals[I_T_FINAL];
	double C_MIN = ps->Reals[I_C_MIN];
	int N_OPEN_MAX = ps->Integers[I_N_RYR_FIDELITY];
	double OUTPUT_GRID_INTERVAL = ps->Reals[I_OUTPUT_GRID_INTERVAL];
	double OUTPUT_STATES_INTERVAL = ps->Reals[I_OUTPUT_STATES_INTERVAL];
	double ca_ss_max = 0;

	//LCC protocol
	int bLCC_Open = ps->Integers[I_FLAG_LCC];
	int seed = 93843 + ps->Integers[I_RAND_SEED_0] + sd->device*ps->Integers[I_RAND_ADD_A] + sd->ensemble_index*ps->Integers[I_RAND_ADD_B];
	if (ps->Integers[I_FLAG_CLOCK_SEED]) {
		seed += (int)time(NULL);
	}
	srand(seed);
	double T_LCC_Close = -ps->Reals[I_LCC_DURATION]*log(((double)rand())/((double)RAND_MAX));
	if (bLCC_Open) {
		fprintf(stdout,"LCC will be closed at %g ms.\n",T_LCC_Close);
	}

	//Channel gating fast-forward option
	//This assumes the system is at steady-state so that Ca2+ concentrations are not changing
	//Simulates only channel gating until a channel opens or T_FINAL is reached
	//Useful for accelerating gain simulations at -40mV where openings are rare
	if (ps->Integers[I_GATING_FFWD]) {
		fprintf(stdout,"Using accelerated steady-state gating...\n");
		int bDone = 0;
		while (!bDone) {

			UpdateChannelGating( dt, &dds, &dgs);
			t += dt;
			dt_last = dt;

			hipMemcpy(sd->RyR_States, dgs.RyR_States, sizeof(int)*dgs.N_RyR, hipMemcpyDeviceToHost);
			hipMemcpy(sd->LCC_States, dgs.LCC_States, sizeof(int)*dgs.N_LCC, hipMemcpyDeviceToHost);
			hipMemcpy(sd->LCCV_States, dgs.LCCV_States, sizeof(int)*dgs.N_LCC, hipMemcpyDeviceToHost);
			hipDeviceSynchronize();
			N_RyR_Open = 0;
			for (int i = 0; i < sd->N_RyR; i++) {
				N_RyR_Open += sd->RyR_States[i];
			}
			int n_open_lcc = 0;
			for (int i = 0; i < sd->N_LCC; i++) {
				if (sd->LCCV_States[i] == 2 && (sd->LCC_States[i] == 6 || sd->LCC_States[i] == 12)) {
					n_open_lcc++;
				}
			}

			//Check if a channel opened or we've reached T_FINAL
			//fprintf(stdout,"RyR open = %d, LCC_open = %d, t = %f\n",N_RyR_Open,n_open_lcc,t);
			if (N_RyR_Open > 0 || n_open_lcc > 0 || t > T_FINAL) {
				bDone = 1;
			}

			//Output states if not finished
			if (!bDone) {
				//Output grid and states
				if (OUTPUT_GRID_FLAG && floor(t/OUTPUT_GRID_INTERVAL) > floor((t-dt_last)/OUTPUT_GRID_INTERVAL)) {

					//Copy data from device
					hipDeviceSynchronize();
					for (int i = 0; i < dds.N_States; i++) {
						hipMemcpy(sd->Grid.States[i],dds.States + (i*dds.N_Ele), sizeof(double)*dds.N_Ele, hipMemcpyDeviceToHost);
					}
					hipDeviceSynchronize();
					printf("Writing grid file (gating accelerated)  (t = %f, #%d).\n",t,output_counter);
					WriteToGridFiles(output_counter, t, &sd->Grid, sd, ps);
					output_counter++;
				}
				if (floor(t/OUTPUT_STATES_INTERVAL) > floor((t-dt_last)/OUTPUT_STATES_INTERVAL)) {
					//Copy data from device
					hipDeviceSynchronize();
					for (int i = 0; i < dds.N_States; i++) { //Note: grid values only needed if calculating total Ca2+
						hipMemcpy(sd->Grid.States[i],dds.States + (i*dds.N_Ele), sizeof(double)*dds.N_Ele, hipMemcpyDeviceToHost);
					}
					hipMemcpy(sd->Global_Vars, dgs.Global_Vars, sizeof(double)*N_GLOBAL_VARS, hipMemcpyDeviceToHost);
					hipDeviceSynchronize();
					printf("Writing to states file (gating accelerated) (t = %f, LCC_Open = %d, RyR_open = %d).\n",t,n_open_lcc,N_RyR_Open);
					WriteToStateFiles(t, sd, ps);
				}
			}

			//Voltage clamp protocol
			if (ps->Integers[I_FLAG_V_CLAMP]) {
				if (t <= ps->Reals[I_T_CLAMP]) {
					sd->Global_Vars[GLOBAL_VM] = ps->Reals[I_V_CLAMP];
				} else {
					sd->Global_Vars[GLOBAL_VM] = ps->Reals[I_DEFAULT_VM];
				}
				hipMemcpy(&dgs.Global_Vars[GLOBAL_VM], &sd->Global_Vars[GLOBAL_VM], sizeof(double), hipMemcpyHostToDevice);
			}
		}
	}

	//Main time loop
	while (t <= T_FINAL && (!FLAG_FIDELITY_SIM || ( (bLCC_Open || N_RyR_Open > 0 || ca_ss_max > C_MIN) && N_RyR_Open < N_OPEN_MAX ))) {

		//LCC protocol
		if (ps->Integers[I_FLAG_LCC] && bLCC_Open && t >= T_LCC_Close ) {
			bLCC_Open = 0;
			sd->LCC_States[ps->Integers[I_LCC_INDEX]] = DEFAULT_STATE_LCC;
			hipMemcpy(&dgs.LCC_States[ps->Integers[I_LCC_INDEX]], &sd->LCC_States[ps->Integers[I_LCC_INDEX]], sizeof(int), hipMemcpyHostToDevice);
		}

		//Voltage clamp protocol
		if (ps->Integers[I_FLAG_V_CLAMP]) {
			if (t <= ps->Reals[I_T_CLAMP]) {
				sd->Global_Vars[GLOBAL_VM] = ps->Reals[I_V_CLAMP];
			} else {
				sd->Global_Vars[GLOBAL_VM] = ps->Reals[I_DEFAULT_VM];
			}
			hipMemcpy(&dgs.Global_Vars[GLOBAL_VM], &sd->Global_Vars[GLOBAL_VM], sizeof(double), hipMemcpyHostToDevice);
		}

		//RyR Hold Option

		//RyR Hold Option
		if (ps->Integers[I_INIT_RYR] != -1 && t < ps->Reals[I_INIT_RYR_HOLD]) {
			hipMemcpy(&sd->RyR_States[ps->Integers[I_INIT_RYR]], &dgs.RyR_States[ps->Integers[I_INIT_RYR]], sizeof(int), hipMemcpyDeviceToHost);
			if (!sd->RyR_States[ps->Integers[I_INIT_RYR]]) {
			    sd->RyR_States[ps->Integers[I_INIT_RYR]] = 1;
				hipMemcpy(&dgs.RyR_States[ps->Integers[I_INIT_RYR]], &sd->RyR_States[ps->Integers[I_INIT_RYR]], sizeof(int), hipMemcpyHostToDevice);
			}
		}

		/*if (t > 2.0) {
			for (int i = 0; i < sd->N_RyR; i++) sd->RyR_States[i] = 0;
			hipMemcpy(&dgs.RyR_States[ps->Integers[I_INIT_RYR]], &sd->RyR_States[ps->Integers[I_INIT_RYR]], sizeof(int), hipMemcpyHostToDevice);
		}*/

		//Output states
		if (OUTPUT_GRID_FLAG && floor(t/OUTPUT_GRID_INTERVAL) > floor((t-dt_last)/OUTPUT_GRID_INTERVAL)) {
			
			//Copy data from device
			hipDeviceSynchronize();
			for (int i = 0; i < dds.N_States; i++) {
				hipMemcpy(sd->Grid.States[i],dds.States + (i*dds.N_Ele), sizeof(double)*dds.N_Ele, hipMemcpyDeviceToHost);
			}
			hipDeviceSynchronize();
			printf("Writing grid file (t = %f, #%d).\n",t,output_counter);
			WriteToGridFiles(output_counter, t, &sd->Grid, sd, ps);
			output_counter++;
		}

		if (floor(t/OUTPUT_STATES_INTERVAL) > floor((t-dt_last)/OUTPUT_STATES_INTERVAL)) {

			//Copy data from device
			hipDeviceSynchronize();
			for (int i = 0; i < dds.N_States; i++) { //Note: grid values only needed if calculating total Ca2+
				hipMemcpy(sd->Grid.States[i],dds.States + (i*dds.N_Ele), sizeof(double)*dds.N_Ele, hipMemcpyDeviceToHost);
			}
			hipMemcpy(sd->RyR_States, dgs.RyR_States, sizeof(int)*dgs.N_RyR, hipMemcpyDeviceToHost);
			hipMemcpy(sd->LCC_States, dgs.LCC_States, sizeof(int)*dgs.N_LCC, hipMemcpyDeviceToHost);
			hipMemcpy(sd->LCCV_States, dgs.LCCV_States, sizeof(int)*dgs.N_LCC, hipMemcpyDeviceToHost);
			hipMemcpy(sd->Global_Vars, dgs.Global_Vars, sizeof(double)*N_GLOBAL_VARS, hipMemcpyDeviceToHost);
			hipDeviceSynchronize();

			N_RyR_Open = 0;
			ca_ss_max = 0;
			for (int i = 0; i < sd->N_RyR; i++) {
				N_RyR_Open += sd->RyR_States[i];
				double c = sd->Grid.States[INDEX_CA][sd->RyR_Ele[i]];
				if (c > ca_ss_max) {
					ca_ss_max = c;
				}
			}

			printf("Writing to states file (t = %f, N_open = %d, Ca_SS_Max = %g).\n",t,N_RyR_Open,ca_ss_max);
			WriteToStateFiles(t, sd, ps);
		}

		//Fast explicit step routine
		ExplicitStep(dt, &dds, &dgs, ps);

		//Increment time step
		t += dt;
		dt_last = dt;

	}

	//Stop CUDA profiler
	hipProfilerStop();

	//Timer
	clock_t clock_end = clock();

	//Output final state
	hipDeviceSynchronize();
	for (int i = 0; i < dds.N_States; i++) {
		hipMemcpy(sd->Grid.States[i],dds.States + (i*dds.N_Ele), sizeof(double)*dds.N_Ele, hipMemcpyDeviceToHost);
	}
	hipMemcpy(sd->RyR_States, dgs.RyR_States, sizeof(int)*dgs.N_RyR, hipMemcpyDeviceToHost);
	hipMemcpy(sd->LCC_States, dgs.LCC_States, sizeof(int)*dgs.N_LCC, hipMemcpyDeviceToHost);
	hipMemcpy(sd->LCCV_States, dgs.LCCV_States, sizeof(int)*dgs.N_LCC, hipMemcpyDeviceToHost);
	hipMemcpy(sd->Global_Vars, dgs.Global_Vars, sizeof(double)*N_GLOBAL_VARS, hipMemcpyDeviceToHost);
	hipMemcpy(sd->R, dgs.R, sizeof(double)*dgs.N_Channels, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	printf("Writing grid files (t = %f, #%d).\n",t,output_counter);
	WriteToGridFiles(output_counter, t, &sd->Grid, sd, ps);
	printf("Writing state files (t = %f).\n",t);
	WriteToStateFiles(t, sd, ps);

	//Write time for simulation
	printf("Simulation completed in %f secs.\n",((float)clock_end-clock_start)/CLOCKS_PER_SEC);

	//Free memory
	FreeDomainStruct(&dds);
	FreeGlobalStruct(&dgs);
	hipDeviceSynchronize();


}

void ExplicitStep(double dt, DeviceDomainStruct* dds, DeviceGlobalStruct* dgs, ParamStruct* ps) {

	int N_Ele = dds->N_Ele;
	int block_size = 256;
	int n_blocks_ele = N_Ele/block_size + (N_Ele%block_size == 0?0:1);
	int n_blocks_addmult = ((N_Ele*dds->N_States)/block_size) + ((N_Ele*dds->N_States)%block_size == 0?0:1);
	int block_size_gv = 8;
	int n_blocks_gv = N_GLOBAL_VARS/block_size_gv + (N_GLOBAL_VARS%block_size_gv == 0?0:1);

	//Initialize global variable delta vector to zero
	resetVectorToZero<<< n_blocks_gv, block_size_gv >>> (dgs->GV_src1, N_GLOBAL_VARS);

	//Diffusion step
	/*for (int i = 0; i < dds->N_States; i++) {
		if (ps->Diff[i] > 0) { //Diffusible states

			//Know this is synchronous as long as the block size is >64KB
			hipMemcpy(thrust::raw_pointer_cast(&dds->Array_2[0]), &dds->States[i*N_Ele], sizeof(double)*N_Ele, hipMemcpyDeviceToDevice);

			cusp::multiply(dds->A[i],dds->Array_2,dds->Array_1[i]);

		} else { //Non-diffusible states
			resetVectorToZero<<< n_blocks_ele, block_size >>> (thrust::raw_pointer_cast(&dds->Array_1[i][0]), N_Ele);
		}

	}*/

	/*for (int i = 0; i < dds->N_States; i++) {
		resetVectorToZero<<< n_blocks_ele, block_size >>> (thrust::raw_pointer_cast(&dds->Array_1[i][0]), N_Ele);
	}*/

	//Buffering reactions
	UpdateReactionTerms<<< n_blocks_ele, block_size >>>(N_Ele,
													dds->States,
													dds->Array_1,
													dgs->Global_Vars,
													dds->aij,
													dds->nij,
													dds->TropC, dds->V0, dds->TTSurfaceArea, dds->SRSurfaceArea, dds->Boundary_Rates,
													dds->bJSR, dds->Vol_JSR);

	//Update Channel Ca fluxes
	UpdateRyRFlux<<< 1, dgs->N_RyR >>>(&dds->States[INDEX_CA*N_Ele],
						&dds->Array_1[INDEX_CA*N_Ele],
						dgs->Global_Vars,
						dds->V0,
						dgs->N_RyR, dgs->RyR_States, dgs->RyR_Ele,
						dgs->RyR_JSR_Ele);
	UpdateLCCFlux<<< 1, dgs->N_LCC >>>(&dds->States[INDEX_CA*N_Ele],
						&dds->Array_1[INDEX_CA*N_Ele],
						dgs->Global_Vars, dds->V0,
						dgs->N_LCC, dgs->LCC_States, dgs->LCCV_States, dgs->LCC_Ele);

	//Fast buffering (must be performed last)
	UpdateFastBuffer<<< n_blocks_ele, block_size >>>(N_Ele,&dds->States[INDEX_CA*N_Ele], &dds->Array_1[INDEX_CA*N_Ele], dds->TTSurfaceArea,dds->V0);

	//Update new states
	/*for (int k = 0; k < dds->N_States; k++) {
		addMultVectors<<< n_blocks_ele, block_size >>> (&dds->States[k*N_Ele], &dds->Array_1[k*N_Ele], dt, N_Ele);
	}*/
	addMultVectors<<< n_blocks_addmult, block_size >>> (dds->States, dds->Array_1, dt, N_Ele*dds->N_States);
	addMultVectors<<< n_blocks_gv, block_size_gv >>> (dgs->Global_Vars, dgs->GV_src1, dt, N_GLOBAL_VARS);

	//Do channel gating
	if (!ps->Integers[I_FLAG_NO_GATING]) {
		UpdateChannelGating( dt, dds, dgs);
	}

	if ( hipSuccess != hipGetLastError() )
		fprintf(stderr,"*****************Cuda Error during step computations!\n");

}

void UpdateChannelGating(double dt, DeviceDomainStruct* dds, DeviceGlobalStruct* dgs) {

	//Do channel gating
	int N_Ele = dds->N_Ele;
	UpdateRyRStates <<< 1, dgs->N_RyR >>> ( dt, dgs->N_RyR, dgs->RyR_States,
											&dds->States[INDEX_CA*N_Ele], dgs->RyR_Ele,
											dgs->randStates, dgs->R, dgs->Global_Vars, dgs->RyR_Neighb,
											dgs->RyR_JSR_Ele);
	UpdateLCCStates <<< 1, dgs->N_LCC >>> ( dt, dgs->N_LCC, dgs->N_RyR, dgs->LCC_States, dgs->LCCV_States,
											dgs->LCC_Ele, &dds->States[INDEX_CA*N_Ele],
											dgs->randStates, dgs->R, dgs->Global_Vars);

}


void InitializeDeviceConstants(ParamStruct* ps) {

	hipMemcpyToSymbol(HIP_SYMBOL(PARAMS_DEVICE),(void*)ps->Reals,sizeof(double)*N_PARAMETERS_REAL,0,hipMemcpyHostToDevice);
}

void FreeDomainStruct(DeviceDomainStruct* dds) {

	//delete[] dds->A;
	hipFree(dds->States);
	hipFree(dds->Boundary_Rates);
	hipFree(dds->TropC);
	hipFree(dds->bJSR);
	hipFree(dds->V0);
	hipFree(dds->TTSurfaceArea);
	hipFree(dds->SRSurfaceArea);

}

void FreeGlobalStruct(DeviceGlobalStruct* dgs) {

	hipFree(dgs->Global_Vars);
	hipFree(dgs->GV_src1);
	hipFree(dgs->RyR_States);
	hipFree(dgs->RyR_Ele);
	hipFree(dgs->RyR_JSR_Ele);
	hipFree(dgs->RyR_Neighb);
	hipFree(dgs->LCC_States);
	hipFree(dgs->LCCV_States);
	hipFree(dgs->LCC_Ele);
	hipFree(dgs->R);
	hipFree(dgs->randStates);

}
