#include "hip/hip_runtime.h"
/*
 * InitFuncs.cpp
 *
 *  Created on: Jun 27, 2012
 *      Author: mwalker
 */

#include <stdio.h>
#include <iostream>
#include <map>
#include <vector>
#include <stdexcept>
#include <xercesc/parsers/XercesDOMParser.hpp>
#include <xercesc/dom/DOM.hpp>
#include <xercesc/sax/HandlerBase.hpp>
#include <xercesc/util/XMLString.hpp>
#include <xercesc/util/PlatformUtils.hpp>
#include "DeviceStructs.h"

using namespace std;
using namespace xercesc;

//Reads parameters
void ReadParameters(ParamStruct* ps, const char* param_file) {

	//Assign default parameters
	ps->Integers[I_OUTPUT_GRID_FLAG] = 0;
	ps->Integers[I_FLAG_FIDELITY_SIM] = 0;
	ps->Integers[I_FLAG_CLOCK_SEED] = 1;
	ps->Integers[I_RAND_SEED_0] = 38376427;
	ps->Integers[I_RAND_ADD_A] = 12783;
	ps->Integers[I_RAND_ADD_B] = 83;
	ps->Integers[I_INITIALIZE_FROM_FILE] = 0;
	ps->Integers[I_CUDA_DEVICE] = 0;
	ps->Integers[I_FLAG_SECONDARY_GRADIENT] = 0;
	ps->Integers[I_FLAG_NO_FLUX_BOUNDARY] = 1;
	ps->Integers[I_N_RYR_FIDELITY] = 1000;
	ps->Integers[I_FLAG_LCC] = 0;
	ps->Integers[I_LCC_INDEX] = 0;
	ps->Integers[I_FLAG_V_CLAMP] = 0;
	ps->Integers[I_FLAG_NO_GATING] = 0;
	ps->Integers[I_GATING_FFWD] = 0;
	ps->Integers[I_FLAG_RYR_OUT] = 0;
	ps->Integers[I_N_SIMS] = 1;
	ps->Integers[I_INIT_RYR] = -1;

	ps->Reals[I_T_FINAL] = 500;
	ps->Reals[I_T_STEP] = 12e-6;
	ps->Reals[I_OUTPUT_STATES_INTERVAL] = 0.01;
	ps->Reals[I_OUTPUT_GRID_INTERVAL] = 1;
	ps->Reals[I_C_MIN] = 1.0;
	ps->Reals[I_D_CA] = 0.25;
	ps->Reals[I_D_CA_JSR] = 0.25;
	ps->Reals[I_D_ATP] = 0.14;
	ps->Reals[I_D_CMDN] = 0.025;
	ps->Reals[I_D_TRPN] = 0;
	ps->Reals[I_D_CSQN] = 0;
	ps->Reals[I_D_DYE] = 0.042;
	ps->Reals[I_D_EGTA] = 0;
	ps->Reals[I_C_0_CA] = 0.1;
	ps->Reals[I_C_0_CA_JSR] = 1000;
	ps->Reals[I_DEFAULT_VM] = -80;
	ps->Reals[I_DEFAULT_CA_NSR] = 1000;
	ps->Reals[I_CA_0] = 2000;
	ps->Reals[I_B_TOT_ATP] = 455;
	ps->Reals[I_K_OFF_ATP] = 45;
	ps->Reals[I_K_ON_ATP] = 0.225;
	ps->Reals[I_B_TOT_CMDN] = 24;
	ps->Reals[I_K_OFF_CMDN] = 0.238;
	ps->Reals[I_K_ON_CMDN] = 0.034;
	ps->Reals[I_B_TOT_TRPN] = 70;
	ps->Reals[I_K_OFF_TRPN] = 0.02;
	ps->Reals[I_K_ON_TRPN] = 0.039;
	ps->Reals[I_B_TOT_SL_HI] = 1.6e-13;
	ps->Reals[I_K_D_SL_HI] = 13;
	ps->Reals[I_B_TOT_SL_LO] = 0;
	ps->Reals[I_K_D_SL_LO] = 1100;
	ps->Reals[I_B_TOT_DYE] = 50;
	ps->Reals[I_K_OFF_DYE] = 0.110;
	ps->Reals[I_K_ON_DYE] = 0.10;
	ps->Reals[I_B_T_JSR] = 30000;
	ps->Reals[I_K_M_JSR] = 638;
	ps->Reals[I_V_REFILL] = 0.095;
	ps->Reals[I_V_RYR] = 3e-8;
	ps->Reals[I_V_CELL] = 25.84e3;
	ps->Reals[I_K_ON_CSQN] = 0.10;
	ps->Reals[I_K_OFF_CSQN] = 63.8;
	ps->Reals[I_RYR_ETA] = 2.1;
	ps->Reals[I_RYR_A_STAR] = 0;
	ps->Reals[I_RYR_EPS_CC] = -0.92;
	ps->Reals[I_RYR_EPS_OO] = -0.85;
	ps->Reals[I_RYR_K_PLUS] = 0.1107e-3;
	ps->Reals[I_RYR_K_MINUS] = 0.500;
	ps->Reals[I_RYR_PHI_M] = 4.0;
	ps->Reals[I_RYR_PHI_B] = 1500;
	ps->Reals[I_K_D_i] = 910;
	ps->Reals[I_K_D_SR] = 2240;
	ps->Reals[I_A_P] = 150;
	ps->Reals[I_LCC_DURATION] = 0.5;
	ps->Reals[I_T_CLAMP] = 200;
	ps->Reals[I_V_CLAMP] = 0;
	ps->Reals[I_INIT_RYR_HOLD] = 0;

	sprintf(ps->Chars[I_FILE_BASE],"mesh");
	sprintf(ps->Chars[I_OUTPUT_STATE],"output");
	sprintf(ps->Chars[I_OUTPUT_GRID],"output");
	sprintf(ps->Chars[I_PARAM_TITLE],"Untitled");

	//Initialize XML parser
	try {
        XMLPlatformUtils::Initialize();
    }
    catch (const XMLException& toCatch) {
        char* message = XMLString::transcode(toCatch.getMessage());
        cout << "Error during initialization! :\n"
             << message << "\n";
        XMLString::release(&message);
        return;
    }

    XercesDOMParser* parser = new XercesDOMParser();
    parser->setValidationScheme(XercesDOMParser::Val_Always);
    parser->setDoNamespaces(true);    // optional

    ErrorHandler* errHandler = (ErrorHandler*) new HandlerBase();
    parser->setErrorHandler(errHandler);

	//Parse XML file
	fprintf(stdout,"Parsing parameter file %s...\n",param_file);
    char* xmlFile = (char*)param_file;

    try {
        parser->parse(xmlFile);
    }
    catch (const XMLException& toCatch) {
        char* message = XMLString::transcode(toCatch.getMessage());
        cout << "Exception message is: \n"
             << message << "\n";
        XMLString::release(&message);
        return;
    }
    catch (const DOMException& toCatch) {
        char* message = XMLString::transcode(toCatch.msg);
        cout << "Exception message is: \n"
             << message << "\n";
        XMLString::release(&message);
        return;
    }
    catch (...) {
        cout << "Unexpected Exception \n" ;
        return;
    }

    DOMDocument* xmlDoc = parser->getDocument();

    XMLCh* TAG_PARAMETER = XMLString::transcode("parameter");
    XMLCh* TAG_SYMBOL = XMLString::transcode("symbol");
    XMLCh* TAG_VALUE = XMLString::transcode("value");

    DOMNodeList* paramElements = xmlDoc->getElementsByTagName(TAG_PARAMETER);
    XMLSize_t nodeCount = paramElements->getLength();

    if (nodeCount == 0) {
    	throw(std::runtime_error("No parameters were found!"));
    }

    for (XMLSize_t xx = 0; xx < nodeCount; ++xx) {
    	DOMNode* currentNode = paramElements->item(xx);
    	if (currentNode->getNodeType() && currentNode->getNodeType() == DOMNode::ELEMENT_NODE) {
    		DOMElement* currentElement = dynamic_cast< xercesc::DOMElement* >(currentNode);
			DOMNodeList* children_param = currentElement->getChildNodes();
			XMLSize_t nodeCount_param = children_param->getLength();
			char* symbol;
			char* value;

			for (XMLSize_t i = 0; i < nodeCount_param; ++i) {
				DOMNode* currentNode_param = children_param->item(i);
				if (currentNode_param->getNodeType() && currentNode_param->getNodeType() == DOMNode::ELEMENT_NODE) {
					DOMElement* currentElement_param = dynamic_cast< xercesc::DOMElement* >(currentNode_param);
					if (XMLString::equals(currentElement_param->getTagName(), TAG_SYMBOL)) {
						symbol = XMLString::transcode(currentElement_param->getTextContent());
					} else if (XMLString::equals(currentElement_param->getTagName(), TAG_VALUE)) {
						value = XMLString::transcode(currentElement_param->getTextContent());
					}
				}
			}

			cout << "symbol = " << symbol << ", value = " << value << endl;

			if (!strcmp(symbol,"flag_noflux")) {
				if (!strcmp(value,"true")) {
					ps->Integers[I_FLAG_NO_FLUX_BOUNDARY] = 1;
				} else {
					ps->Integers[I_FLAG_NO_FLUX_BOUNDARY] = 0;
				}
			} else if (!strcmp(symbol,"flag_nogating")) {
				if (!strcmp(value,"true")) {
					ps->Integers[I_FLAG_NO_GATING] = 1;
				} else {
					ps->Integers[I_FLAG_NO_GATING] = 0;
				}
			} else if (!strcmp(symbol,"flag_ffwd")) {
				if (!strcmp(value,"true")) {
					ps->Integers[I_GATING_FFWD] = 1;
				} else {
					ps->Integers[I_GATING_FFWD] = 0;
				}
			} else if (!strcmp(symbol,"flag_use_clock")) {
				if (!strcmp(value,"true")) {
					ps->Integers[I_FLAG_CLOCK_SEED] = 1;
				} else {
					ps->Integers[I_FLAG_CLOCK_SEED] = 0;
				}
			} else if (!strcmp(symbol,"flag_output_grid")) {
				if (!strcmp(value,"true")) {
					ps->Integers[I_OUTPUT_GRID_FLAG] = 1;
				} else {
					ps->Integers[I_OUTPUT_GRID_FLAG] = 0;
				}
			} else if (!strcmp(symbol,"protocol")) {
				if (!strcmp(value,"gain")) {
					ps->Integers[I_FLAG_V_CLAMP] = 1;
					ps->Integers[I_FLAG_FIDELITY_SIM] = 0;
				} else if (!strcmp(value,"fidelity")) {
					ps->Integers[I_FLAG_V_CLAMP] = 0;
					ps->Integers[I_FLAG_FIDELITY_SIM] = 1;
				} else {
					ps->Integers[I_FLAG_V_CLAMP] = 0;
					ps->Integers[I_FLAG_FIDELITY_SIM] = 0;
				}
			} else if (!strcmp(symbol,"flag_output_ryr_states")) {
				if (!strcmp(value,"true")) {
					ps->Integers[I_FLAG_RYR_OUT] = 1;
				} else {
					ps->Integers[I_FLAG_RYR_OUT] = 0;
				}
			} else if (!strcmp(symbol,"sims_per_proc")) {
				ps->Integers[I_N_SIMS] = atoi(value);
			} else if (!strcmp(symbol,"ryr_open_init")) {
				ps->Integers[I_INIT_RYR] = atoi(value);
			} else if (!strcmp(symbol,"c_min")) {
				ps->Reals[I_C_MIN] = atof(value);
			} else if (!strcmp(symbol,"ryr_open_time")) {
				ps->Reals[I_INIT_RYR_HOLD] = atof(value);
			} else if (!strcmp(symbol,"ryr_open_max")) {
				ps->Integers[I_N_RYR_FIDELITY] = atoi(value);
			} else if (!strcmp(symbol,"v_clamp")) {
				ps->Reals[I_V_CLAMP] = atof(value);
			} else if (!strcmp(symbol,"t_clamp")) {
				ps->Reals[I_T_CLAMP] = atof(value);
			} else if (!strcmp(symbol,"t_final")) {
				ps->Reals[I_T_FINAL] = atof(value);
			} else if (!strcmp(symbol,"t_step")) {
				ps->Reals[I_T_STEP] = atof(value)*(1e-6); //convert nanosecond to milliseconds
			} else if (!strcmp(symbol,"states_interval")) {
				ps->Reals[I_OUTPUT_STATES_INTERVAL] = atof(value);
			} else if (!strcmp(symbol,"grid_interval")) {
				ps->Reals[I_OUTPUT_GRID_INTERVAL] = atof(value);
			} else if (!strcmp(symbol,"seed_base")) {
				ps->Integers[I_RAND_SEED_0] = atoi(value);
			} else if (!strcmp(symbol,"seed_A")) {
				ps->Integers[I_RAND_ADD_A] = atoi(value);
			} else if (!strcmp(symbol,"seed_B")) {
				ps->Integers[I_RAND_ADD_B] = atoi(value);
			} else if (!strcmp(symbol,"CA_I")) {
				ps->Reals[I_C_0_CA] = atof(value);
			} else if (!strcmp(symbol,"CA_SR")) {
				ps->Reals[I_C_0_CA_JSR] = atof(value);
				ps->Reals[I_DEFAULT_CA_NSR] = atof(value);
			} else if (!strcmp(symbol,"CA_O")) {
				ps->Reals[I_CA_0] = atof(value);
			} else if (!strcmp(symbol,"V_M")) {
				ps->Reals[I_DEFAULT_VM] = atof(value);
			} else if (!strcmp(symbol,"D_CA")) {
				ps->Reals[I_D_CA] = atof(value);
			} else if (!strcmp(symbol,"D_CAJSR")) {
				ps->Reals[I_D_CA_JSR] = atof(value);
			} else if (!strcmp(symbol,"D_ATP")) {
				ps->Reals[I_D_ATP] = atof(value);
			} else if (!strcmp(symbol,"D_CMDN")) {
				ps->Reals[I_D_CMDN] = atof(value);
			} else if (!strcmp(symbol,"D_DYE")) {
				ps->Reals[I_D_DYE] = atof(value);
			} else if (!strcmp(symbol,"B_TOT_ATP")) {
				ps->Reals[I_B_TOT_ATP] = atof(value);
			} else if (!strcmp(symbol,"K_OFF_ATP")) {
				ps->Reals[I_K_OFF_ATP] = atof(value);
			} else if (!strcmp(symbol,"K_ON_ATP")) {
				ps->Reals[I_K_ON_ATP] = atof(value);
			} else if (!strcmp(symbol,"B_TOT_CMDN")) {
				ps->Reals[I_B_TOT_CMDN] = atof(value);
			} else if (!strcmp(symbol,"K_OFF_CMDN")) {
				ps->Reals[I_K_OFF_CMDN] = atof(value);
			} else if (!strcmp(symbol,"K_ON_CMDN")) {
				ps->Reals[I_K_ON_CMDN] = atof(value);
			} else if (!strcmp(symbol,"B_TOT_TRPN")) {
				ps->Reals[I_B_TOT_TRPN] = atof(value);
			} else if (!strcmp(symbol,"K_OFF_TRPN")) {
				ps->Reals[I_K_OFF_TRPN] = atof(value);
			} else if (!strcmp(symbol,"K_ON_TRPN")) {
				ps->Reals[I_K_ON_TRPN] = atof(value);
			} else if (!strcmp(symbol,"B_TOT_SL")) {
				ps->Reals[I_B_TOT_SL_HI] = atof(value);
			} else if (!strcmp(symbol,"K_D_SL")) {
				ps->Reals[I_K_D_SL_HI] = atof(value);
			} else if (!strcmp(symbol,"B_TOT_DYE")) {
				ps->Reals[I_B_TOT_DYE] = atof(value);
			} else if (!strcmp(symbol,"K_OFF_DYE")) {
				ps->Reals[I_K_OFF_DYE] = atof(value);
			} else if (!strcmp(symbol,"K_ON_DYE")) {
				ps->Reals[I_K_ON_DYE] = atof(value);
			} else if (!strcmp(symbol,"B_TOT_CSQN")) {
				ps->Reals[I_B_T_JSR] = atof(value);
			} else if (!strcmp(symbol,"K_OFF_CSQN")) {
				ps->Reals[I_K_OFF_CSQN] = atof(value);
			} else if (!strcmp(symbol,"K_ON_CSQN")) {
				ps->Reals[I_K_ON_CSQN] = atof(value);
			} else if (!strcmp(symbol,"V_REFILL")) {
				ps->Reals[I_V_REFILL] = atof(value);
			} else if (!strcmp(symbol,"V_RYR")) {
				ps->Reals[I_V_RYR] = atof(value);
			} else if (!strcmp(symbol,"RYR_ETA")) {
				ps->Reals[I_RYR_ETA] = atof(value);
			} else if (!strcmp(symbol,"RYR_A_STAR")) {
				ps->Reals[I_RYR_A_STAR] = atof(value);
			} else if (!strcmp(symbol,"RYR_EPS_CC")) {
				ps->Reals[I_RYR_EPS_CC] = atof(value);
			} else if (!strcmp(symbol,"RYR_EPS_OO")) {
				ps->Reals[I_RYR_EPS_OO] = atof(value);
			} else if (!strcmp(symbol,"RYR_K_MINUS")) {
				ps->Reals[I_RYR_K_MINUS] = atof(value);
			} else if (!strcmp(symbol,"RYR_K_PLUS")) {
				ps->Reals[I_RYR_K_PLUS] = atof(value);
			} else if (!strcmp(symbol,"RYR_PHI_N")) {
				ps->Reals[I_RYR_PHI_M] = atof(value);
			} else if (!strcmp(symbol,"RYR_PHI_KD")) {
				ps->Reals[I_RYR_PHI_B] = atof(value);
			} else if (!strcmp(symbol,"K_D_I")) {
				ps->Reals[I_K_D_i] = atof(value);
			} else if (!strcmp(symbol,"K_D_SR")) {
				ps->Reals[I_K_D_SR] = atof(value);
			} else if (!strcmp(symbol,"A_P")) {
				ps->Reals[I_A_P] = atof(value);
			} else if (!strcmp(symbol,"paramset_title")) {
				sprintf(ps->Chars[I_PARAM_TITLE],"%s",value);
    		}
    	}
    }

    delete parser;
    delete errHandler;

	//Initialize parameter arrays

	//Diffusion coefficients
	ps->Diff[0] = ps->Reals[I_D_CA];
	ps->Diff[1] = ps->Reals[I_D_ATP];
	ps->Diff[2] = ps->Reals[I_D_CMDN];
	ps->Diff[3] = ps->Reals[I_D_TRPN];
	ps->Diff[4] = ps->Reals[I_D_CSQN];
	ps->Diff[5] = ps->Reals[I_D_DYE];

	//Default dynamic global variable values
	ps->Default_Globals[0] = ps->Reals[I_DEFAULT_VM];
	ps->Default_Globals[1] = ps->Reals[I_DEFAULT_CA_NSR];

	for (int i = 0; i < N_PARAMETERS_INT; i++) {
		fprintf(stdout,"Integer Parameter[%d] = %d\n",i,ps->Integers[i]);
	}
	for (int i = 0; i < N_PARAMETERS_REAL; i++) {
		fprintf(stdout,"Real Parameter[%d] = %g\n",i,ps->Reals[i]);
	}
	for (int i = 0; i < N_PARAMETERS_CHAR; i++) {
		fprintf(stdout,"Char Parameter[%d] = %s\n",i,ps->Chars[i]);
	}


}

//Reads grid structure from files
void ReadGrid(SimData* sd, GridStruct* gs, ParamStruct* ps) {

	char filename[255];
	sprintf(filename,"%s_tet_properties.txt",ps->Chars[I_FILE_BASE]);

	FILE* pFile;
	pFile = fopen(filename,"r");
	char line[255];

	fprintf(stdout,"Reading element property file %s...\n",filename);
	if (pFile != NULL) {

		//Read number of elements
		if (!fscanf(pFile,"%d\n",&(gs->N_Ele) )) {
			fprintf(stderr,"Error reading element property : number of elements\n");
		}

		//Allocate space for element property arrays
		try {
			gs->Domain = new int[gs->N_Ele];
			gs->V0 = new double[gs->N_Ele];
			gs->TTSurfaceArea = new double[gs->N_Ele];
			gs->SRSurfaceArea = new double[gs->N_Ele];
			gs->TropC = new int[gs->N_Ele];
		} catch (std::bad_alloc& ba)
		{
			fprintf(stderr,"bad_alloc caught: %s\n",ba.what());
		}

		//Skip column labels
		if (fgets(line,255,pFile) == NULL) {
			fprintf(stderr,"Error skipping first header line in element property file.\n");
		}

		//Read properties
		for (int i = 0; i < gs->N_Ele; i++) {
			double j1,j2,j3;
			if (!fscanf(pFile,"%lg %lg %lg %lg %lg %lg %d %d\n",&j1,&j2,&j3,
						&(gs->V0[i]),&(gs->TTSurfaceArea[i]),&(gs->SRSurfaceArea[i]),&(gs->TropC[i]),&(gs->Domain[i]) )) {
				fprintf(stderr,"Error reading element property data at line %d.\n",i+3);
			}
		}

		fclose(pFile);
	} else {
		fprintf(stderr,"Error opening element properties file: %s\n",filename);
	}

	//Read face file
	sprintf(filename,"%s_face_properties.txt",ps->Chars[I_FILE_BASE]);
	fprintf(stdout,"Reading face property file %s...\n",filename);
	pFile = fopen(filename,"r");

	if (pFile != NULL) {

		//Read number of faces
		if (!fscanf(pFile,"%d\n",&(gs->N_Face) )) {
			fprintf(stderr,"Error reading face property : number of faces.\n");
		}

		//Allocate space for triangle property arrays
		try {
			gs->Ele_1 = new int[gs->N_Face];
			gs->Ele_2 = new int[gs->N_Face];
			gs->FaceType = new int[gs->N_Face];
			gs->Face_C = new double[gs->N_Face];
			gs->dXi = new double[gs->N_Face];
			gs->Area_f = new double[gs->N_Face];
			gs->Afx = new double[gs->N_Face];
			gs->Afy = new double[gs->N_Face];
			gs->Afz = new double[gs->N_Face];
			gs->eXix = new double[gs->N_Face];
			gs->eXiy = new double[gs->N_Face];
			gs->eXiz = new double[gs->N_Face];
			gs->r1x = new double[gs->N_Face];
			gs->r1y = new double[gs->N_Face];
			gs->r1z = new double[gs->N_Face];
			gs->r2x = new double[gs->N_Face];
			gs->r2y = new double[gs->N_Face];
			gs->r2z = new double[gs->N_Face];
		} catch (std::bad_alloc& ba)
		{
			fprintf(stderr,"bad_alloc caught: %s\n",ba.what());
		}

		//Skip column labels
		if (fgets(line,255,pFile) == NULL) {
			fprintf(stderr,"Error skipping second header line in face property file.\n");
		}

		//Read properties
		for (int i = 0; i < gs->N_Face; i++) {

			double j1,j2,j3,j4,j5,j6;
			if ( !fscanf(pFile,"%d %d %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %lg %d\n",
					&(gs->Ele_1[i]),&(gs->Ele_2[i]),
					&j1,&j2,&j3, //do not load cell normals
					&j4,&j5,&j6, //do not load cell centers
					&(gs->Area_f[i]),
					&(gs->Afx[i]),&(gs->Afy[i]),&(gs->Afz[i]),
					&(gs->r1x[i]),&(gs->r1y[i]),&(gs->r1z[i]),
					&(gs->r2x[i]),&(gs->r2y[i]),&(gs->r2z[i]),
					&(gs->eXix[i]),&(gs->eXiy[i]),&(gs->eXiz[i]),
					&(gs->dXi[i]),
					&(gs->Face_C[i]),
					&(gs->FaceType[i])) ) {
				fprintf(stderr,"Error reading face data at line %d.\n",i+3);
			}

			//Convert from geometry interface type to simulation interface type
			switch (gs->FaceType[i]) {
			case IN_FACE_DEFAULT:
				gs->FaceType[i] = FACE_DEFAULT;
				break;
			case IN_FACE_BOUNDARY:
				if (ps->Integers[I_FLAG_NO_FLUX_BOUNDARY]) {
					gs->FaceType[i] = FACE_NOFLUX;
				} else {
					gs->FaceType[i] = FACE_BOUND;
				}
				break;
			case IN_FACE_TT:
				gs->FaceType[i] = FACE_NOFLUX;
				break;
			case IN_FACE_JSR:
				gs->FaceType[i] = FACE_NOFLUX;
				break;
			default:
				fprintf(stderr,"Warning: undefined input face type %d.\n",gs->FaceType[i]);
				break;
			}
		}
		fclose(pFile);
	} else {
		fprintf(stderr,"Error opening face properties file: %s\n",filename);
	}

	sprintf(filename,"%s.node",ps->Chars[I_FILE_BASE]);
	fprintf(stdout,"Reading node file %s...\n",filename);
	pFile = fopen(filename,"r");
	if (pFile != NULL) {

		//Read number of nodes
		if (!fscanf(pFile,"%d",&(gs->N_Nodes) )) {
			fprintf(stderr,"Error node file - number of nodes\n");
		}

		//Allocate space for element property arrays
		try {
			gs->Node_X = new double[gs->N_Nodes];
			gs->Node_Y = new double[gs->N_Nodes];
			gs->Node_Z = new double[gs->N_Nodes];
		} catch (std::bad_alloc& ba)
		{
			fprintf(stderr,"bad_alloc caught: %s\n",ba.what());
		}

		//Skip column labels
		if (fgets(line,255,pFile) == NULL) {
			fprintf(stderr,"Error skipping first header line in element node file.\n");
		}

		//Read properties
		for (int i = 0; i < gs->N_Nodes; i++) {
			int j1;
			if (!fscanf(pFile,"%d %lg %lg %lg\n",&j1,&(gs->Node_X[i]),&(gs->Node_Y[i]),&(gs->Node_Z[i])) ) {
				fprintf(stderr,"Error reading element property data at line %d.\n",i+3);
			}
		}

		fclose(pFile);
	} else {
		fprintf(stderr,"Error opening element properties file: %s\n",filename);
	}

	sprintf(filename,"%s.ele",ps->Chars[I_FILE_BASE]);
	fprintf(stdout,"Reading cell-node file %s...\n",filename);
	pFile = fopen(filename,"r");
	if (pFile != NULL) {

		//Skip column labels
		if (fgets(line,255,pFile) == NULL) {
			fprintf(stderr,"Error skipping first header line in element node file.\n");
		}

		//Allocate space for element property arrays
		try {
			gs->Ele_Nodes = new int[gs->N_Ele*4];
		} catch (std::bad_alloc& ba)
		{
			fprintf(stderr,"bad_alloc caught: %s\n",ba.what());
		}

		//Read properties
		for (int i = 0; i < gs->N_Ele; i++) {
			int j1,j2;
			int idx = 4*i;
			if (!fscanf(pFile,"%d %d %d %d %d %d\n",&j1,&(gs->Ele_Nodes[idx]),&(gs->Ele_Nodes[idx+1]),&(gs->Ele_Nodes[idx+2]),&(gs->Ele_Nodes[idx+3]),&j2) ) {
				fprintf(stderr,"Error reading element property data at line %d.\n",i+3);
			}

			//Convert to 0-based indexes
			gs->Ele_Nodes[idx]--;
			gs->Ele_Nodes[idx+1]--;
			gs->Ele_Nodes[idx+2]--;
			gs->Ele_Nodes[idx+3]--;
		}

		fclose(pFile);
	} else {
		fprintf(stderr,"Error opening element properties file: %s\n",filename);
	}

	sprintf(filename,"%s_lcc.txt",ps->Chars[I_FILE_BASE]);
	fprintf(stdout,"Reading LCC file %s...\n",filename);
	pFile = fopen(filename,"r");
	if (pFile != NULL) {

		//Read number of channels
		if (!fscanf(pFile,"%d\n",&(sd->N_LCC) )) {
			fprintf(stderr,"Error node file - number of channels\n");
		}

		//Allocate space for element property arrays
		try {
			sd->LCC_Ele = new int[sd->N_LCC];
		} catch (std::bad_alloc& ba)
		{
			fprintf(stderr,"bad_alloc caught: %s\n",ba.what());
		}

		//Read properties
		for (int i = 0; i < sd->N_LCC; i++) {
			if (!fscanf(pFile,"%d\n",&(sd->LCC_Ele[i]) )) {
				fprintf(stderr,"Error reading LCC property data at line %d.\n",i+3);
			}
		}
		fclose(pFile);
	} else {
		fprintf(stderr,"Error opening file: %s\n",filename);
	}

	sprintf(filename,"%s_ryr.txt",ps->Chars[I_FILE_BASE]);
	fprintf(stdout,"Reading RyR file %s...\n",filename);
	pFile = fopen(filename,"r");
	if (pFile != NULL) {

		//Read number of channels
		if (!fscanf(pFile,"%d\n",&(sd->N_RyR) )) {
			fprintf(stderr,"Error node file - number of channels\n");
		}

		//Allocate space for element property arrays
		try {
			sd->RyR_Ele = new int[sd->N_RyR];
			sd->RyR_JSR_Ele = new int[sd->N_RyR];
			sd->RyR_Neighb = new int[sd->N_RyR*4];
		} catch (std::bad_alloc& ba)
		{
			fprintf(stderr,"bad_alloc caught: %s\n",ba.what());
		}

		//Read properties
		for (int i = 0; i < sd->N_RyR; i++) {
			if (!fscanf(pFile,"%d %d %d %d %d %d\n",&(sd->RyR_Ele[i]),&(sd->RyR_JSR_Ele[i]),&(sd->RyR_Neighb[i*4+0]),&(sd->RyR_Neighb[i*4+1]),&(sd->RyR_Neighb[i*4+2]),&(sd->RyR_Neighb[i*4+3]) )) {
				fprintf(stderr,"Error reading RyR property data at line %d.\n",i+3);
			}
		}
		fclose(pFile);
	} else {
		fprintf(stderr,"Error opening file: %s\n",filename);
	}

	for (int i = 0; i < sd->N_RyR; i++) {
		for (int j = i+1; j < sd->N_RyR; j++) {
			if (sd->RyR_Ele[i] == sd->RyR_Ele[j] || sd->RyR_JSR_Ele[i] == sd->RyR_JSR_Ele[j]) {
				fprintf(stderr,"**Warning: RyRs %d and %d located in same element. Parallel race conditions may exist.\n",i,j);
			}
		}
	}

	for (int i = 0; i < sd->N_LCC; i++) {
		for (int j = i+1; j < sd->N_LCC; j++) {
			if (sd->LCC_Ele[i] == sd->LCC_Ele[j]) {
				fprintf(stderr,"**Warning: LCCs %d and %d located in same element. Parallel race conditions may exist.\n",i,j);
			}
		}
	}

	//Add total number of channels
	sd->N_Channels = sd->N_RyR + sd->N_LCC;

	fprintf(stdout,"Done reading input mesh. # Faces = %d, # Elements = %d\n",gs->N_Face,gs->N_Ele);

	fprintf(stdout,"Optimizing grid numbering structure...\n");
	vector<int> ele_1; //lists of adjacent elements
	vector<int> ele_2;
	for (int i = 0; i < sd->Grid.N_Face; i++) {
		if (gs->Ele_2[i] >= 0) {
			ele_1.push_back(gs->Ele_1[i]);
			ele_2.push_back(gs->Ele_2[i]);
		}
	}

	//The next section re-orders elements to improve memory access times
	//Simulation gets up to a ~10-50% speed increase depending on mesh size

	//Modified version of Reverse Cuthill-McKee (RCM) algorithm
	vector<int> vertices;
	vector<int> visited;
	vertices.push_back(0); //Starting node
	while (!vertices.empty()) {
		int v = vertices[0];
		vertices.erase(vertices.begin());
		visited.push_back(v);

		//Find unvisited neighbors of v
		for (int i = 0; i < ele_1.size(); i++) {
			if (ele_1[i] == v || ele_2[i] == v) {
				int neighb;
				if (ele_1[i] == v) {
					neighb = ele_2[i];
				} else {
					neighb = ele_1[i];
				}
				int bSkip = 0;
				for (int j = 0; j < vertices.size(); j++) {
					if (vertices[j] == neighb) {
						bSkip = 1;
						break;
					}
				}
				if (!bSkip) {
					for (int j = 0; j < visited.size(); j++) {
						if (visited[j] == neighb) {
							bSkip = 1;
							break;
						}
					}
					if (!bSkip) {
						vertices.push_back(neighb);
						//ele_1.erase(ele_1.begin() + i);
						//ele_2.erase(ele_2.begin() + i);
					}
				}
			}
		}
	}

	//Remap elements
	fprintf(stdout,"Remapping elements...\n");
	int* Domain = new int[gs->N_Ele];
	double* V0 = new double[gs->N_Ele];
	double* TTSurfaceArea = new double[gs->N_Ele];
	double* SRSurfaceArea = new double[gs->N_Ele];
	int* TropC = new int[gs->N_Ele];
	int* Ele_Nodes = new int[gs->N_Ele*4];
	int* Ele_1 = new int[gs->N_Face];
	int* Ele_2 = new int[gs->N_Face];
	int* RyR_Ele = new int[sd->N_RyR];
	int* RyR_JSR_Ele = new int[sd->N_RyR];
	int* LCC_Ele = new int[sd->N_LCC];

	memcpy(Domain,gs->Domain,gs->N_Ele*sizeof(int));
	memcpy(V0,gs->V0,gs->N_Ele*sizeof(double));
	memcpy(TTSurfaceArea,gs->TTSurfaceArea,gs->N_Ele*sizeof(double));
	memcpy(SRSurfaceArea,gs->SRSurfaceArea,gs->N_Ele*sizeof(double));
	memcpy(TropC,gs->TropC,gs->N_Ele*sizeof(int));
	memcpy(Ele_Nodes,gs->Ele_Nodes,4*gs->N_Ele*sizeof(int));
	memcpy(Ele_1,gs->Ele_1,gs->N_Face*sizeof(int));
	memcpy(Ele_2,gs->Ele_2,gs->N_Face*sizeof(int));
	memcpy(RyR_Ele,sd->RyR_Ele,sd->N_RyR*sizeof(int));
	memcpy(RyR_JSR_Ele,sd->RyR_JSR_Ele,sd->N_RyR*sizeof(int));
	memcpy(LCC_Ele,sd->LCC_Ele,sd->N_LCC*sizeof(int));

	for (int i = 0; i < gs->N_Ele; i++) {
		gs->Domain[i] = Domain[visited[i]];
		gs->V0[i] = V0[visited[i]];
		gs->TTSurfaceArea[i] = TTSurfaceArea[visited[i]];
		gs->SRSurfaceArea[i] = SRSurfaceArea[visited[i]];
		gs->TropC[i] = TropC[visited[i]];
		gs->Ele_Nodes[4*i+0] = Ele_Nodes[4*visited[i]+0];
		gs->Ele_Nodes[4*i+1] = Ele_Nodes[4*visited[i]+1];
		gs->Ele_Nodes[4*i+2] = Ele_Nodes[4*visited[i]+2];
		gs->Ele_Nodes[4*i+3] = Ele_Nodes[4*visited[i]+3];

		for (int j = 0; j < gs->N_Face; j++) {
			if (Ele_1[j] == visited[i]) {
				gs->Ele_1[j] = i;
			}
			if (Ele_2[j] == visited[i]) {
				gs->Ele_2[j] = i;
			}
		}
		for (int j = 0; j < sd->N_RyR; j++) {
			if (RyR_Ele[j] == visited[i]) {
				sd->RyR_Ele[j] = i;
			}
		}
		for (int j = 0; j < sd->N_RyR; j++) {
			if (RyR_JSR_Ele[j] == visited[i]) {
				sd->RyR_JSR_Ele[j] = i;
			}
		}
		for (int j = 0; j < sd->N_LCC; j++) {
			if (LCC_Ele[j] == visited[i]) {
				sd->LCC_Ele[j] = i;
			}
		}
	}

	fprintf(stdout,"Computing matrix properties...\n");
	//Compute maximum and mean distance from a neighbor
	int max_dist = 0;
	double mean_dist = 0;
	for (int i = 0; i < gs->N_Face; i++) {
		if (gs->Ele_2[i] >= 0) {
			int dist = abs(gs->Ele_1[i]-gs->Ele_2[i]);
			mean_dist += (double)dist;
			if (dist > max_dist) {
				max_dist = dist;
			}
		}
	}
	mean_dist /= gs->N_Face;
	fprintf(stdout,"\tMax / Mean neighbor distance: %d / %g\n",max_dist,mean_dist);

	delete[] Domain, V0, TTSurfaceArea, SRSurfaceArea, TropC, Ele_Nodes, Ele_1, Ele_2, RyR_Ele, LCC_Ele;

}

void ReadGridStates(GridStruct* gs, ParamStruct* ps) {

	char filename[255];
	sprintf(filename,"%s_init_states.txt",ps->Chars[I_FILE_BASE]);

	//Read Grid File
	fprintf(stdout,"Reading grid initial conditions file %s...\n",filename);
	FILE* pFile;
	pFile = fopen(filename,"r");
	char line[255];

	if (pFile != NULL) {

		//Skip header
		fgets(line,255,pFile);
		fgets(line,255,pFile);

		//Get number of states
		int temp1, temp2;
		if (!fscanf(pFile,"%lf %d %d\n",&temp1, &temp2, &gs->N_States)) {
			fprintf(stderr,"Error reading grid header, could not get number of states.\n");
			exit(0);
		}

		fgets(line,255,pFile);
		fgets(line,255,pFile);

		//Allocate memory
		gs->States = new double*[gs->N_States];
		for (int i = 0; i < gs->N_States; i++) {
			gs->States[i] = new double[gs->N_Ele];
		}

		//Read grid data
		for (int k = 0; k < gs->N_States; k++) {
			for (int i = 0; i < gs->N_Ele; i++) {
				if (!fscanf(pFile,"%lf ",&(gs->States[k][i]))) {
					fprintf(stderr,"Error reading grid initial conditions data for state %d, cell %d.\n",k, i);
				}
			}
			fgets(line,255,pFile);
		}

		fclose(pFile);
	} else {
		fprintf(stderr,"Error: Could not open grid initial conditions file %s.\n",filename);
		exit(0);
	}


}

void InitializeDefaultGrid(SimData* sd, ParamStruct* ps) {

	////////////////////////
	//Cytosolic grid
	////////////////////////

	GridStruct* gs = &sd->Grid;

	//Use states specified in parameters.h
	gs->N_States = N_STATES;

	//Allocate grid memory
	gs->States = new double*[gs->N_States];
	for (int i = 0; i < gs->N_States; i++) {
		gs->States[i] = new double[gs->N_Ele];
	}
	gs->Boundaries = new double[gs->N_Ele];

	//Initialize default values
	ResetDefaultGridStates(sd,ps);

}

void ResetDefaultGridStates(SimData* sd, ParamStruct* ps) {

	GridStruct* gs = &sd->Grid;

	//Calculate equilibrium concentrations given fixed Ca2+
	double ca0 = ps->Reals[I_C_0_CA];
	double ca0_jsr = ps->Reals[I_C_0_CA_JSR];
	double Kd, btot;

	//TroponinC
	Kd = ps->Reals[I_K_OFF_TRPN]/ps->Reals[I_K_ON_TRPN];
	double c_trpn  = ca0 * ps->Reals[I_B_TOT_TRPN] / (Kd + ca0);

	//Calsequestrin
	Kd = ps->Reals[I_K_OFF_CSQN]/ps->Reals[I_K_ON_CSQN];
	double c_csqn  = ca0_jsr * ps->Reals[I_B_T_JSR] / (Kd + ca0_jsr);

	//ATP
	Kd = ps->Reals[I_K_OFF_ATP]/ps->Reals[I_K_ON_ATP];
	double c_atp  = ca0 * ps->Reals[I_B_TOT_ATP] / (Kd + ca0);

	//Calmodulin
	Kd = ps->Reals[I_K_OFF_CMDN]/ps->Reals[I_K_ON_CMDN];
	double c_cmdn  = ca0 * ps->Reals[I_B_TOT_CMDN] / (Kd + ca0);

	//Dye
	Kd = ps->Reals[I_K_OFF_DYE]/ps->Reals[I_K_ON_DYE];
	double c_dye  = ca0 * ps->Reals[I_B_TOT_DYE] / (Kd + ca0);

	//Set boundary conditions
	gs->Boundaries[INDEX_CA] = ca0;
	gs->Boundaries[INDEX_TRPN] = c_trpn;
	gs->Boundaries[INDEX_CSQN] = 0;
	gs->Boundaries[INDEX_ATP] = c_atp;
	gs->Boundaries[INDEX_CMDN] = c_cmdn;
	gs->Boundaries[INDEX_DYE] = c_dye;

	//Initialize grid values
	for (int j = 0; j < gs->N_Ele; j++) {

		if (gs->Domain[j] == DOMAIN_CYTO) {
			gs->States[INDEX_CA][j] = ca0;
			gs->States[INDEX_TRPN][j] = c_trpn*gs->TropC[j];
			gs->States[INDEX_ATP][j] = c_atp;
			gs->States[INDEX_CMDN][j] = c_cmdn;

			/*
			//Sarcolemmal binding sites must be calculated for each cell
			btot = gs->TTSurfaceArea[j]*B_TOT_SL / (gs->V0[j]*(1e-15));
			Kd = K_OFF_SL/K_ON_SL;
			double c_sl  = ca0 * btot / (Kd + ca0);
			gs->States[INDEX_SL][j] = c_sl;
			*/

			gs->States[INDEX_DYE][j] = c_dye;

		} else if (gs->Domain[j] == DOMAIN_JSR) {
			gs->States[INDEX_CA][j] = ca0_jsr;
			gs->States[INDEX_TRPN][j] = 0;
			gs->States[INDEX_CSQN][j] = c_csqn;
			gs->States[INDEX_ATP][j] = 0;
			gs->States[INDEX_CMDN][j] = 0;
			gs->States[INDEX_DYE][j] = 0;
		}
	}
}

void InitializeDefaultChannels(SimData* sd) {

	sd->LCC_States = new int[sd->N_LCC];
	sd->LCCV_States = new int[sd->N_LCC];

	//Initialize residuals
	sd->R = new double[sd->N_Channels];

	//Reset to default values
	ResetDefaultChannels(sd);

}

void ResetDefaultChannels(SimData* sd) {

	//Initialize channel states
	for (int i = 0; i < sd->N_LCC; i++) {
		sd->LCC_States[i] = DEFAULT_STATE_LCC;
		sd->LCCV_States[i] = DEFAULT_STATE_LCCV;
	}

	sd->RyR_States = new int[sd->N_RyR];
	for (int i = 0; i < sd->N_RyR; i++) {
		sd->RyR_States[i] = DEFAULT_STATE_RYR;
	}
}

void ResetGlobals(SimData* sd, ParamStruct* ps) {

	//Initialize default global variables
	for (int i = 0; i < N_GLOBAL_VARS; i++) {
		sd->Global_Vars[i] = ps->Default_Globals[i];
	}
}

__global__ void FindDiagonals(int N_Ele, int* diag, int n_cols, int* col_indices) {

	//Note the columns indices are stored in column-major order
	for (int j = 0; j < n_cols; j++) {
		for (int k = 0; k < N_Ele; k++) {
			if (col_indices[j*N_Ele + k] == k) {
				diag[k] = j*N_Ele + k;
			}
		}
	}
}

void InitializeDomainStruct(DeviceDomainStruct* dds, GridStruct* gs, int bSecondaryGradient, ParamStruct* ps) {

	//Create sparse matrices
	//fprintf(stdout,"Initializing domain structure...\n");
	//dds->A = new sp_type[gs->N_States];
	InitializeLeastSquares(dds,gs,bSecondaryGradient,ps);

	if ( hipSuccess != hipGetLastError() )
		fprintf(stderr,"*****************Cuda Error during InitializeLeastSquares()!\n");

	//fprintf(stdout,"Allocating memory for dense vectors...\n");
	//Allocate dense vector arrays
	//dds->Array_2 = cusp::array1d<double,cusp::device_memory>(gs->N_Ele);


	if ( hipSuccess != hipGetLastError() )
		fprintf(stderr,"*****************Cuda Error!\n");
	//Grid buffer
	//fprintf(stdout,"Allocating memory for buffer vectors...\n");
	/*dds->Array_1 = new cusp::array1d<double,cusp::device_memory>[gs->N_States];
	for (int i = 0; i < gs->N_States; i++) {
		dds->Array_1[i] = cusp::array1d<double,cusp::device_memory>(gs->N_Ele);
	}*/

	if ( hipSuccess != hipGetLastError() )
		fprintf(stderr,"*****************Cuda Error!\n");

	//fprintf(stdout,"Computing boundary rates...\n");
	//Boundary fluxes for diffusible states
	double** boundary_rates = new double*[gs->N_States];
	for (int k = 0; k < gs->N_States; k++) {
		boundary_rates[k] = new double[gs->N_Ele];
		for (int i = 0; i < gs->N_Ele; i++) {
			boundary_rates[k][i] = 0;
		}
		for (int i = 0; i < gs->N_Face; i++) {
			int e = i;
			int tri1 = gs->Ele_1[e];
			if (gs->FaceType[e] == FACE_BOUND) {
				boundary_rates[k][tri1] += ps->Diff[k] * (gs->Face_C[e]) * gs->Boundaries[k] / gs->V0[tri1];
			}
		}
	}

	//JSR labels and volume
	int* bJSR = new int[gs->N_Ele];
	dds->Vol_JSR = 0;
	for (int i = 0; i < gs->N_Ele; i++) {
		bJSR[i] = gs->Domain[i] == DOMAIN_JSR ? 1 : 0;
		if (gs->Domain[i] == DOMAIN_JSR) {
			dds->Vol_JSR += gs->V0[i];
		}
	}

	//Number of grid states
	dds->N_States = gs->N_States;
	//Number of grid elements
	dds->N_Ele = gs->N_Ele;


	if ( hipSuccess != hipGetLastError() )
		fprintf(stderr,"*****************Cuda Error initializing boundaries and JSR!\n");
	//fprintf(stdout,"Allocating device memory...\n");
	//Allocate arrays on device
	hipMalloc((void**) &dds->Array_1, sizeof(double)*(dds->N_States)*dds->N_Ele);
	hipMalloc((void**) &dds->States, sizeof(double)*(dds->N_States)*dds->N_Ele);
	hipMalloc((void**) &dds->Boundary_Rates, sizeof(double)*(dds->N_States)*dds->N_Ele);
	hipMalloc((void**) &dds->TropC, sizeof(int)*dds->N_Ele);
	hipMalloc((void**) &dds->bJSR, sizeof(int)*dds->N_Ele);
	hipMalloc((void**) &dds->V0, sizeof(double)*dds->N_Ele);
	hipMalloc((void**) &dds->TTSurfaceArea, sizeof(double)*dds->N_Ele);
	hipMalloc((void**) &dds->SRSurfaceArea, sizeof(double)*dds->N_Ele);
	hipMalloc((void**) &dds->aij, sizeof(double)*5*dds->N_Ele);
	hipMalloc((void**) &dds->nij, sizeof(int)*4*dds->N_Ele);


	if ( hipSuccess != hipGetLastError() )
		fprintf(stderr,"*****************Cuda Error hipMalloc'ing space!\n");

	//Copy data to device
	//fprintf(stdout,"Copying to device memory...\n");
	for (int i = 0; i < dds->N_States; i++) {
		hipMemcpy(dds->States + (i*gs->N_Ele), gs->States[i], sizeof(double)*dds->N_Ele, hipMemcpyHostToDevice);
		hipMemcpy(dds->Boundary_Rates + (i*gs->N_Ele), boundary_rates[i], sizeof(double)*dds->N_Ele, hipMemcpyHostToDevice);
	}
	hipMemcpy(dds->TropC, gs->TropC, sizeof(int)*dds->N_Ele, hipMemcpyHostToDevice);
	hipMemcpy(dds->bJSR, bJSR, sizeof(int)*dds->N_Ele, hipMemcpyHostToDevice);
	hipMemcpy(dds->V0, gs->V0, sizeof(double)*dds->N_Ele, hipMemcpyHostToDevice);
	hipMemcpy(dds->TTSurfaceArea, gs->TTSurfaceArea, sizeof(double)*dds->N_Ele, hipMemcpyHostToDevice);
	hipMemcpy(dds->SRSurfaceArea, gs->SRSurfaceArea, sizeof(double)*dds->N_Ele, hipMemcpyHostToDevice);
	hipMemcpy(dds->aij, gs->aij, sizeof(double)*5*dds->N_Ele, hipMemcpyHostToDevice);
	hipMemcpy(dds->nij, gs->nij, sizeof(int)*4*dds->N_Ele, hipMemcpyHostToDevice);

	if ( hipSuccess != hipGetLastError() )
		fprintf(stderr,"*****************Cuda Error copying data to device!\n");

	//Free memory
	//fprintf(stdout,"Freeing memory...\n");
	for (int k = 0; k < gs->N_States; k++) {
		delete[] boundary_rates[k];
	}
	delete[] boundary_rates;
	delete[] bJSR;

}

void InitializeLeastSquares(DeviceDomainStruct* dds, GridStruct* gs, int bSecondaryGradient, ParamStruct* ps) {

	//Sparse matrix for gradient calculation

	if (!bSecondaryGradient) {

		//Create sparse matrix
		fprintf(stdout,"Computing sparse matrix elements (primary gradient only)...\n");
		/*for (int i = 0; i < gs->N_States; i++) {
			dds->A[i] = InitializeExplicitSparseMatrixFast(gs,i,0,0,bSecondaryGradient,ps);
		}*/
		InitializeExplicitSparseMatrixFast(gs,0,0,0,bSecondaryGradient,ps);

	} else {
		fprintf(stdout,"Computing sparse matrix elements (secondary gradient)...\n");
		//Least-squares matrices (estimate = MLS*DLS)
		double** MLS = new double*[gs->N_Ele];
		for (int i = 0; i < gs->N_Ele; i++) {
			MLS[i] = new double[12];
		}

		//2D array of faces of each cell
		int** NLS = new int*[gs->N_Ele];
		for (int i = 0; i < gs->N_Ele; i++) {
			NLS[i] = new int[4];
			for (int j = 0; j < 4; j++) {
				NLS[i][j] = -1;
			}
		}

		//Calculate least-squares matrices
		int* numSet = new int[gs->N_Ele];
		for (int i = 0; i < gs->N_Ele; i++) {
			numSet[i] = 0;
		}

		double** M0 = new double*[gs->N_Ele];
		for (int i = 0; i < gs->N_Ele; i++) {
			M0[i] = new double[12];
		}

		double** M1 = new double*[gs->N_Ele];
		for (int i = 0; i < gs->N_Ele; i++) {
			M1[i] = new double[9];
		}

		double** W = new double*[gs->N_Ele];
		for (int i = 0; i < gs->N_Ele; i++) {
			W[i] = new double[4];
		}

		//Calculate M0=M matrix
		int tri1,tri2,idx;
		double x1,x2,x3,w;
		for (int i = 0; i < gs->N_Face; i++) {
			tri1 = gs->Ele_1[i];
			tri2 = gs->Ele_2[i];

			if (gs->FaceType[i] == FACE_DEFAULT) {
				x1 = gs->eXix[i]*gs->dXi[i];
				x2 = gs->eXiy[i]*gs->dXi[i];
				x3 = gs->eXiz[i]*gs->dXi[i];
			} else {
				x1 = gs->r1x[i];
				x2 = gs->r1y[i];
				x3 = gs->r1z[i];
			}

			idx = 3*numSet[tri1];

			//Weighted regression
			w = (x1*x1 + x2*x2 + x3*x3);
			//w = 1;
			if (w == 0) {
				fprintf(stderr,"Warning: infinite weight, assigning to 1e6.\n");
				w = 1e6;
			}
			W[tri1][numSet[tri1]] = 1/w;

			M0[tri1][idx] = x1/w;
			M0[tri1][idx+1] = x2/w;
			M0[tri1][idx+2] = x3/w;
			NLS[tri1][numSet[tri1]] = i;
			numSet[tri1]++;


			if (gs->FaceType[i] != FACE_BOUND) {

				if (tri2 >= 0) {
					if (gs->FaceType[i] == FACE_DEFAULT) {
						x1 = -gs->eXix[i]*gs->dXi[i];
						x2 = -gs->eXiy[i]*gs->dXi[i];
						x3 = -gs->eXiz[i]*gs->dXi[i];
					} else { //else interface
						x1 = gs->r2x[i];
						x2 = gs->r2y[i];
						x3 = gs->r2z[i];
					}

					idx = 3*numSet[tri2];

					//Weighted regression
					w = (x1*x1 + x2*x2 + x3*x3);
					//w = 1;
					if (w == 0) {
						fprintf(stderr,"Warning: infinite weight, assigning to 1e6.\n");
						w = 1e6;
					}
					W[tri2][numSet[tri2]] = 1/w;

					M0[tri2][idx] = x1/w;
					M0[tri2][idx+1] = x2/w;
					M0[tri2][idx+2] = x3/w;
					NLS[tri2][numSet[tri2]] = i;
					numSet[tri2]++;
				}
			}
		}

		//Calculate M1 = M0^T * M0
		for (int i = 0; i < gs->N_Ele; i++) {
			M1[i][0] = M0[i][0]*M0[i][0] + M0[i][3]*M0[i][3] + M0[i][6]*M0[i][6] + M0[i][9]*M0[i][9];
			M1[i][1] = M0[i][0]*M0[i][1] + M0[i][3]*M0[i][4] + M0[i][6]*M0[i][7] + M0[i][9]*M0[i][10];
			M1[i][2] = M0[i][0]*M0[i][2] + M0[i][3]*M0[i][5] + M0[i][6]*M0[i][8] + M0[i][9]*M0[i][11];
			M1[i][3] = M0[i][1]*M0[i][0] + M0[i][4]*M0[i][3] + M0[i][7]*M0[i][6] + M0[i][10]*M0[i][9];
			M1[i][4] = M0[i][1]*M0[i][1] + M0[i][4]*M0[i][4] + M0[i][7]*M0[i][7] + M0[i][10]*M0[i][10];
			M1[i][5] = M0[i][1]*M0[i][2] + M0[i][4]*M0[i][5] + M0[i][7]*M0[i][8] + M0[i][10]*M0[i][11];
			M1[i][6] = M0[i][2]*M0[i][0] + M0[i][5]*M0[i][3] + M0[i][8]*M0[i][6] + M0[i][11]*M0[i][9];
			M1[i][7] = M0[i][2]*M0[i][1] + M0[i][5]*M0[i][4] + M0[i][8]*M0[i][7] + M0[i][11]*M0[i][10];
			M1[i][8] = M0[i][2]*M0[i][2] + M0[i][5]*M0[i][5] + M0[i][8]*M0[i][8] + M0[i][11]*M0[i][11];
		}

		//Calculate inverse of M1
		double a,b,c,d,e,f,g,h,k, det;
		double A, B, C, D, E, F, G, H, K;
		for (int i = 0; i < gs->N_Ele; i++) {
			a = M1[i][0];
			b = M1[i][1];
			c = M1[i][2];
			d = M1[i][3];
			e = M1[i][4];
			f = M1[i][5];
			g = M1[i][6];
			h = M1[i][7];
			k = M1[i][8];
			det = a*e*k + b*f*g + c*d*h - c*e*g - b*d*k - a*f*h;

			if (det==0) {
				fprintf(stderr,"Warning: singular least-square matrix (%d), setting det=1e-6.\n",i);
				det = 1e-6;
			}

			A = e*k - f*h;
			B = f*g - d*k;
			C = d*h - e*g;
			D = c*h - b*k;
			E = a*k - c*g;
			F = g*b - a*h;
			G = b*f - c*e;
			H = c*d - a*f;
			K = a*e - b*d;

			M1[i][0] = A/det;
			M1[i][1] = D/det;
			M1[i][2] = G/det;
			M1[i][3] = B/det;
			M1[i][4] = E/det;
			M1[i][5] = H/det;
			M1[i][6] = C/det;
			M1[i][7] = F/det;
			M1[i][8] = K/det;
		}

		//Calculate MLS = M1^-1 * M0^T
		for (int i = 0; i < gs->N_Ele; i++) {
			MLS[i][0] = M1[i][0]*M0[i][0] + M1[i][1]*M0[i][1] + M1[i][2]*M0[i][2];
			MLS[i][1] = M1[i][0]*M0[i][3] + M1[i][1]*M0[i][4] + M1[i][2]*M0[i][5];
			MLS[i][2] = M1[i][0]*M0[i][6] + M1[i][1]*M0[i][7] + M1[i][2]*M0[i][8];
			MLS[i][3] = M1[i][0]*M0[i][9] + M1[i][1]*M0[i][10] + M1[i][2]*M0[i][11];
			MLS[i][4] = M1[i][3]*M0[i][0] + M1[i][4]*M0[i][1] + M1[i][5]*M0[i][2];
			MLS[i][5] = M1[i][3]*M0[i][3] + M1[i][4]*M0[i][4] + M1[i][5]*M0[i][5];
			MLS[i][6] = M1[i][3]*M0[i][6] + M1[i][4]*M0[i][7] + M1[i][5]*M0[i][8];
			MLS[i][7] = M1[i][3]*M0[i][9] + M1[i][4]*M0[i][10] + M1[i][5]*M0[i][11];
			MLS[i][8] = M1[i][6]*M0[i][0] + M1[i][7]*M0[i][1] + M1[i][8]*M0[i][2];
			MLS[i][9] = M1[i][6]*M0[i][3] + M1[i][7]*M0[i][4] + M1[i][8]*M0[i][5];
			MLS[i][10] = M1[i][6]*M0[i][6] + M1[i][7]*M0[i][7] + M1[i][8]*M0[i][8];
			MLS[i][11] = M1[i][6]*M0[i][9] + M1[i][7]*M0[i][10] + M1[i][8]*M0[i][11];
			//printf("%d %f %f %f %f %f %f\n",i,MLS[i][0],MLS[i][1],MLS[i][2],MLS[i][3],MLS[i][4],MLS[i][5]);
		}

		//Factor in weights
		for (int i = 0; i < gs->N_Ele; i++) {
			MLS[i][0] *= W[i][0];
			MLS[i][1] *= W[i][1];
			MLS[i][2] *= W[i][2];
			MLS[i][3] *= W[i][3];
			MLS[i][4] *= W[i][0];
			MLS[i][5] *= W[i][1];
			MLS[i][6] *= W[i][2];
			MLS[i][7] *= W[i][3];
			MLS[i][8] *= W[i][0];
			MLS[i][9] *= W[i][1];
			MLS[i][10] *= W[i][2];
			MLS[i][11] *= W[i][3];
		}

		//Create sparse matrix
		/*for (int i = 0; i < gs->N_States; i++) {
			dds->A[i] = InitializeExplicitSparseMatrixFast(gs,i,MLS,NLS,bSecondaryGradient,ps);
		}*/
		InitializeExplicitSparseMatrixFast(gs,0,MLS,NLS,bSecondaryGradient,ps);

		//Free memory
		for (int i = 0; i < gs->N_Ele; i++) {
			delete[] MLS[i];
		}
		delete[] MLS;
		for (int i = 0; i < gs->N_Ele; i++) {
			delete[] NLS[i];
		}
		delete[] NLS;
		delete[] numSet;
		for (int i = 0; i < gs->N_Ele; i++) {
			delete[] M0[i];
		}
		delete[] M0;
		for (int i = 0; i < gs->N_Ele; i++) {
			delete[] M1[i];
		}
		delete[] M1;
		for (int i = 0; i < gs->N_Ele; i++) {
			delete[] W[i];
		}
		delete[] W;
	}
}

//Class used for constructing sparse matrices with hash maps
class Entry2D {
public:
    int i;
    int j;
    int N_Cols;
    bool operator<(const Entry2D &other) const {
        return (((long int)this->i)*(this->N_Cols) + (this->j) < ((long int)other.i)*(this->N_Cols) + (other.j));
    }
    bool operator==(const Entry2D &other) const {
        return (this->i == other.i && this->j == other.j);
    }
};

//int Entry2D::N_Rows = 0; //Should be set to number of cells in code


void InitializeExplicitSparseMatrixFast(GridStruct* gs,int state,double** MLS,int** NLS, int bSecondaryGradient, ParamStruct* ps) {

	//Creates sparse matrix from using a map object, should work better for larger meshes
	//This matrix should be used for explicit time stepping

	int N_Face = gs->N_Face; //Number of edges
	int N_Cells = gs->N_Ele; //Number of control volume elements

	map< Entry2D ,double> valuemap;

	//Primary gradient
	double k1,k2;
	int tri1,tri2;
	Entry2D key;
	key.N_Cols = N_Cells;
	for (int i = 0; i < N_Face; i++) {
		tri1 = gs->Ele_1[i];
		tri2 = gs->Ele_2[i];

		/*double diff = ps->Diff[state];
		if (gs->Domain[tri1] == DOMAIN_JSR) {
			diff = ps->Reals[I_D_CA_JSR];
		}*/
		double diff = 1; //TODO

		k1 = diff * (gs->Face_C[i]) / gs->V0[tri1];

		if (gs->FaceType[i] == FACE_DEFAULT) {

			key.i = tri1;
			key.j = tri2;
			valuemap[key] += k1;

			key.i = tri1;
			key.j = tri1;
			valuemap[key] += -k1;

			k2 = diff * (gs->Face_C[i]) / gs->V0[tri2];

			key.i = tri2;
			key.j = tri1;
			valuemap[key] += k2;

			key.i = tri2;
			key.j = tri2;
			valuemap[key] += -k2;

		} else if (gs->FaceType[i] == FACE_BOUND) {

			key.i = tri1;
			key.j = tri1;
			//valuemap[key] += -k1; //Uncomment to enable diffusion out of domain boundaries

		}
	}

	//Secondary gradient
	if (bSecondaryGradient) {
		map< Entry2D ,double> secondarymap;
		int c0, c1, f, n;
		double g_c1x[5],g_c0x[5];
		double g_c1y[5],g_c0y[5];
		double g_c1z[5],g_c0z[5];
		for (int i = 0; i < N_Face; i++) {
			c0 = gs->Ele_1[i];
			c1 = gs->Ele_2[i];

			double diff = ps->Diff[state];
			if (gs->Domain[c0] == DOMAIN_JSR) {
				diff = ps->Reals[I_D_CA_JSR];
			}

			if (gs->FaceType[i] == FACE_DEFAULT) {

				//X direction
				g_c1x[1] = 0.5 * MLS[c1][0] * diff * (-gs->Afx[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXix[i]));
				g_c1x[2] = 0.5 * MLS[c1][1] * diff * (-gs->Afx[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXix[i]));
				g_c1x[3] = 0.5 * MLS[c1][2] * diff * (-gs->Afx[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXix[i]));
				g_c1x[4] = 0.5 * MLS[c1][3] * diff * (-gs->Afx[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXix[i]));
				g_c1x[0] = -(g_c1x[1] + g_c1x[2] + g_c1x[3] + g_c1x[4]);

				g_c0x[1] = 0.5 * MLS[c0][0] * diff * (-gs->Afx[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXix[i]));
				g_c0x[2] = 0.5 * MLS[c0][1] * diff * (-gs->Afx[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXix[i]));
				g_c0x[3] = 0.5 * MLS[c0][2] * diff * (-gs->Afx[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXix[i]));
				g_c0x[4] = 0.5 * MLS[c0][3] * diff * (-gs->Afx[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXix[i]));
				g_c0x[0] = -(g_c0x[1] + g_c0x[2] + g_c0x[3] + g_c0x[4]);

				//Y direction
				g_c1y[1] = 0.5 * MLS[c1][4] * diff * (-gs->Afy[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiy[i]));
				g_c1y[2] = 0.5 * MLS[c1][5] * diff * (-gs->Afy[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiy[i]));
				g_c1y[3] = 0.5 * MLS[c1][6] * diff * (-gs->Afy[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiy[i]));
				g_c1y[4] = 0.5 * MLS[c1][7] * diff * (-gs->Afy[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiy[i]));
				g_c1y[0] = -(g_c1y[1] + g_c1y[2] + g_c1y[3] + g_c1y[4]);

				g_c0y[1] = 0.5 * MLS[c0][4] * diff * (-gs->Afy[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiy[i]));
				g_c0y[2] = 0.5 * MLS[c0][5] * diff * (-gs->Afy[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiy[i]));
				g_c0y[3] = 0.5 * MLS[c0][6] * diff * (-gs->Afy[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiy[i]));
				g_c0y[4] = 0.5 * MLS[c0][7] * diff * (-gs->Afy[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiy[i]));
				g_c0y[0] = -(g_c0y[1] + g_c0y[2] + g_c0y[3] + g_c0y[4]);

				//Z direction
				g_c1z[1] = 0.5 * MLS[c1][8] * diff * (-gs->Afz[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiz[i]));
				g_c1z[2] = 0.5 * MLS[c1][9] * diff * (-gs->Afz[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiz[i]));
				g_c1z[3] = 0.5 * MLS[c1][10] * diff * (-gs->Afz[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiz[i]));
				g_c1z[4] = 0.5 * MLS[c1][11] * diff * (-gs->Afz[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiz[i]));
				g_c1z[0] = -(g_c1z[1] + g_c1z[2] + g_c1z[3] + g_c1z[4]);

				g_c0z[1] = 0.5 * MLS[c0][8] * diff * (-gs->Afz[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiz[i]));
				g_c0z[2] = 0.5 * MLS[c0][9] * diff * (-gs->Afz[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiz[i]));
				g_c0z[3] = 0.5 * MLS[c0][10] * diff * (-gs->Afz[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiz[i]));
				g_c0z[4] = 0.5 * MLS[c0][11] * diff * (-gs->Afz[i] + (gs->Face_C[i]*gs->dXi[i]*gs->eXiz[i]));
				g_c0z[0] = -(g_c0z[1] + g_c0z[2] + g_c0z[3] + g_c0z[4]);


				//Visit C0's neighbors
				//fprintf(stdout,"%g\n",-g_c0z[0] / gs->V0[c1]);
				for (int j = 0; j < 4; j++) {
					f = NLS[c0][j];
					if (gs->FaceType[f] == FACE_DEFAULT) {
						n = gs->Ele_1[f];
						if (n == c0) {
							n = gs->Ele_2[f];
						}

						key.i = c0;
						key.j = c0;
						valuemap[key] += -(g_c0x[j+1] + g_c0y[j+1] + g_c0z[j+1]) / gs->V0[c0];
						secondarymap[key] += -(g_c0x[j+1] + g_c0y[j+1] + g_c0z[j+1]) / gs->V0[c0];

						key.i = c0;
						key.j = n;
						valuemap[key] += g_c0x[j+1] / gs->V0[c0];
						valuemap[key] += g_c0y[j+1] / gs->V0[c0];
						valuemap[key] += g_c0z[j+1] / gs->V0[c0];
						secondarymap[key] += g_c0x[j+1] / gs->V0[c0];
						secondarymap[key] += g_c0y[j+1] / gs->V0[c0];
						secondarymap[key] += g_c0z[j+1] / gs->V0[c0];

						key.i = c1;
						key.j = c1;
						valuemap[key] -= -(g_c0x[j+1] + g_c0y[j+1] + g_c0z[j+1]) / gs->V0[c1];
						secondarymap[key] -= -(g_c0x[j+1] + g_c0y[j+1] + g_c0z[j+1]) / gs->V0[c1];

						key.i = c1;
						key.j = n;
						valuemap[key] -= g_c0x[j+1] / gs->V0[c1];
						valuemap[key] -= g_c0y[j+1] / gs->V0[c1];
						valuemap[key] -= g_c0z[j+1] / gs->V0[c1];
						secondarymap[key] -= g_c0x[j+1] / gs->V0[c1];
						secondarymap[key] -= g_c0y[j+1] / gs->V0[c1];
						secondarymap[key] -= g_c0z[j+1] / gs->V0[c1];
					}
				}

				//Visit C1's neighbors
				for (int j = 0; j < 4; j++) {
					f = NLS[c1][j];
					if (gs->FaceType[f] == FACE_DEFAULT) {
						n = gs->Ele_1[f];
						if (n == c1) {
							n = gs->Ele_2[f];
						}

						key.i = c0;
						key.j = c0;
						valuemap[key] += -(g_c1x[j+1] + g_c1y[j+1] + g_c1z[j+1]) / gs->V0[c0];
						secondarymap[key] += -(g_c1x[j+1] + g_c1y[j+1] + g_c1z[j+1]) / gs->V0[c0];

						key.i = c0;
						key.j = n;
						valuemap[key] += g_c1x[j+1] / gs->V0[c0];
						valuemap[key] += g_c1y[j+1] / gs->V0[c0];
						valuemap[key] += g_c1z[j+1] / gs->V0[c0];
						secondarymap[key] += g_c1x[j+1] / gs->V0[c0];
						secondarymap[key] += g_c1y[j+1] / gs->V0[c0];
						secondarymap[key] += g_c1z[j+1] / gs->V0[c0];

						key.i = c1;
						key.j = c1;
						valuemap[key] -= -(g_c1x[j+1] + g_c1y[j+1] + g_c1z[j+1]) / gs->V0[c1];
						secondarymap[key] -= -(g_c1x[j+1] + g_c1y[j+1] + g_c1z[j+1]) / gs->V0[c1];

						key.i = c1;
						key.j = n;
						valuemap[key] -= g_c1x[j+1] / gs->V0[c1];
						valuemap[key] -= g_c1y[j+1] / gs->V0[c1];
						valuemap[key] -= g_c1z[j+1] / gs->V0[c1];
						secondarymap[key] -= g_c1x[j+1] / gs->V0[c1];
						secondarymap[key] -= g_c1y[j+1] / gs->V0[c1];
						secondarymap[key] -= g_c1z[j+1] / gs->V0[c1];
					}
				}

			}
		}
/*
		FILE * pFile;
		map<Entry2D,double>::iterator it;
		char file[255];
		sprintf(file,"secondary_matrix_%d.txt",state);
		pFile = fopen (file,"w");
		if (pFile != NULL) {
			for (it = secondarymap.begin(); it != secondarymap.end(); it++) {
				if ((*it).second != 0) {
					fprintf(pFile,"%d %d %g\n",(*it).first.i,(*it).first.j,(*it).second);
				}
			}
			fclose(pFile);
		} else {
			fprintf(stderr,"Warning: could not open matrix file %s.\n", file);
		}
*/
	}


	//Compute coordinate-based data
	int N_Nonzero = 0;
	map<Entry2D,double>::iterator it;
	for (it = valuemap.begin(); it != valuemap.end(); it++) {
		if ((*it).second != 0) {
			N_Nonzero++;
		}
	}
	double* vals = new double[N_Nonzero];
	int* r = new int[N_Nonzero];
	int* c = new int[N_Nonzero];
	int idx = 0;
	for (it = valuemap.begin(); it != valuemap.end(); it++) {
		if ((*it).second != 0) {
			vals[idx] = (*it).second;
			r[idx] = (*it).first.i;
			c[idx] = (*it).first.j;
			//fprintf(stdout,"val[%d] = %g, r[%d] = %d, c[%d] = %d\n",idx,vals[idx],idx,r[idx],idx,c[idx]);
			idx++;
		}
	}

	//Construct custom sparse matrix
	//If an element does not have 4 neighbors, make coefficient 0 and set neighbor to self
	gs->aij = new double[5*gs->N_Ele];
	gs->nij = new int[4*gs->N_Ele];
	int* n_faces = new int[gs->N_Ele];
	for (int i = 0; i < 5*gs->N_Ele; i++) {
		gs->aij[i] = 0;
	}
	for (int i = 0; i < gs->N_Ele; i++) {
		gs->nij[i*4] = i;
		gs->nij[i*4+1] = i;
		gs->nij[i*4+2] = i;
		gs->nij[i*4+3] = i;
	}
	for (int i = 0; i < gs->N_Ele; i++) {
		n_faces[i] = 0;
	}
	for (int i = 0; i < N_Nonzero; i++) {
		int row = r[i];
		int col = c[i];
		if (row == col) {
			gs->aij[5*row] = vals[i];
		} else {
			gs->aij[5*row + 1 + n_faces[row]] = vals[i];
			gs->nij[4*row + n_faces[row]] = col;
			n_faces[row]++;
		}
	}

	delete[] n_faces;

/*
	//First construct coordinate-based matrix since it's easier
	cusp::coo_matrix<int,double,cusp::host_memory> Ac(N_Cells,N_Cells,N_Nonzero);

	for (int i = 0; i < N_Nonzero; i++) {
		Ac.row_indices[i] = r[i];
		Ac.column_indices[i] = c[i];
		Ac.values[i] = vals[i];
	}
*/


	//Check matrix validity
	/*fprintf(stdout,"Verifying matrix validity...\n");
	if (N_Nonzero > 0) {

		//Check validity of the row and column array order
		for (int i = 1; i < N_Nonzero; i++) {
			if (r[i] < r[i-1]) {
				fprintf(stderr,"Error: rows out of order.\n");
				exit(0);
			}
		}

		int current_row = r[0];
		int last_col = c[0];
		for (int i = 1; i < N_Nonzero; i++) {
			if (r[i] > current_row) {
				current_row = r[i];
			} else {
				if (c[i] < last_col) {
					fprintf(stderr,"Error: columns out of order.\n");
					exit(0);
				}
			}
		}

		//Check validity of row and column values
		for (int i = 0; i < N_Nonzero; i++) {
			if (r[i] < 0 || r[i] >= N_Cells) {
				fprintf(stderr,"Error: invalid row r[%d] = %d\n",i,r[i]);
				exit(0);
			}
			if (c[i] < 0 || c[i] >= N_Cells) {
				fprintf(stderr,"Error: invalid column c[%d] = %d\n",i,c[i]);
				exit(0);
			}
		}

		//Check for NaN values
		for (int i = 0; i < N_Nonzero; i++) {
			if (vals[i] != vals[i]) {
				fprintf(stderr,"Error: sparse matrix value vals[%d] = %d\n",i,vals[i]);
				exit(0);
			}
		}
	}
*/

/*
	FILE * pFile;
	if (OUTPUT_GRID_FLAG) {
		char file[255];
		sprintf(file,"sparsematrix_%d.txt",state);
		pFile = fopen (file,"w");
		if (pFile != NULL) {
			//Output grid states.
			//Each state variable's grid is written one at a time
			for (int j = 0; j < N_Nonzero; j++) {
				fprintf(pFile,"%d %d %g\n",r[j],c[j],vals[j]);
			}
			fclose(pFile);
		} else {
			fprintf(stderr,"Warning: could not open matrix file %s.\n", file);
		}
	}
*/

	//fprintf(stdout,"Constructing ELL sparse matrix... ");
	//sp_type Cc(Ac);
	//fprintf(stdout,"done.\n");

	delete[] vals;
	delete[] r;
	delete[] c;

	//Return ELL sparse matrix
	//return Cc;


}

//Device function for initializing PRNG and residuals
__global__ void InitializeRandom(hiprandState * randStates, int seed, double* R, int n_channels) {

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < n_channels) {
		hiprand_init ( seed, idx, 0, &randStates[idx] );

		hiprandState localState = randStates[idx];
		R[idx] = log((double)hiprand_uniform( &localState ));
		randStates[idx] =localState;
	}

}

void InitializeDeviceGlobalStruct(DeviceGlobalStruct* dgs, SimData* sd, ParamStruct* ps) {

	//Initialize variables
	dgs->N_Channels = sd->N_Channels;
	dgs->N_RyR = sd->N_RyR;
	dgs->N_LCC = sd->N_LCC;

	//Allocate space on device
	hipMalloc((void**) &dgs->Global_Vars, sizeof(double)*N_GLOBAL_VARS);
	hipMalloc((void**) &dgs->GV_src1, sizeof(double)*N_GLOBAL_VARS);
	hipMalloc((void**) &dgs->RyR_States, dgs->N_RyR*sizeof(int));
	hipMalloc((void**) &dgs->RyR_Neighb, 4*dgs->N_RyR*sizeof(int));
	hipMalloc((void**) &dgs->RyR_Ele, dgs->N_RyR*sizeof(int));
	hipMalloc((void**) &dgs->RyR_JSR_Ele, dgs->N_RyR*sizeof(int));
	hipMalloc((void**) &dgs->LCC_States, dgs->N_LCC*sizeof(int));
	hipMalloc((void**) &dgs->LCCV_States, dgs->N_LCC*sizeof(int));
	hipMalloc((void**) &dgs->LCC_Ele, dgs->N_LCC*sizeof(int));
	hipMalloc((void**) &dgs->R, sizeof(double)*dgs->N_Channels);

	//Copy data to device
	hipMemcpy(dgs->Global_Vars, sd->Global_Vars, sizeof(double)*N_GLOBAL_VARS, hipMemcpyHostToDevice);
	hipMemcpy(dgs->RyR_States, sd->RyR_States, sizeof(int)*dgs->N_RyR, hipMemcpyHostToDevice);
	hipMemcpy(dgs->RyR_Ele, sd->RyR_Ele, sizeof(int)*dgs->N_RyR, hipMemcpyHostToDevice);
	hipMemcpy(dgs->RyR_JSR_Ele, sd->RyR_JSR_Ele, sizeof(int)*dgs->N_RyR, hipMemcpyHostToDevice);
	hipMemcpy(dgs->RyR_Neighb, sd->RyR_Neighb, 4*sizeof(int)*dgs->N_RyR, hipMemcpyHostToDevice);
	hipMemcpy(dgs->LCC_States, sd->LCC_States, sizeof(int)*dgs->N_LCC, hipMemcpyHostToDevice);
	hipMemcpy(dgs->LCCV_States, sd->LCCV_States, sizeof(int)*dgs->N_LCC, hipMemcpyHostToDevice);
	hipMemcpy(dgs->LCC_Ele, sd->LCC_Ele, sizeof(int)*dgs->N_LCC, hipMemcpyHostToDevice);
	hipMemcpy(dgs->R, sd->R, sizeof(double)*dgs->N_Channels, hipMemcpyHostToDevice);

	//Random numbers
    hipMalloc ( &dgs->randStates, dgs->N_Channels*sizeof( hiprandState ) );

    // setup seeds
	int block_size = 32;
	int n_blocks = dgs->N_Channels/block_size + (dgs->N_Channels%block_size == 0?0:1);
	int seed = ps->Integers[I_RAND_SEED_0] + sd->device*ps->Integers[I_RAND_ADD_A] + sd->ensemble_index*ps->Integers[I_RAND_ADD_B];
	if (ps->Integers[I_FLAG_CLOCK_SEED]) {
		seed += (int)time(NULL);
	}
    InitializeRandom <<< n_blocks, block_size >>> ( dgs->randStates, seed, dgs->R, dgs->N_Channels );
    hipDeviceSynchronize();

	hipMemcpy(sd->R, dgs->R, sizeof(double)*dgs->N_Channels, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

}

void OpenRandomRyR(ParamStruct* ps, SimData* sd) {

	double* RyR_Rates = new double[sd->N_RyR];
	double Rate_Total = 0;

	for (int idx = 0; idx < sd->N_RyR; idx++) {
		int N_Open = 0;
		int N_Closed = 0;
		for (int i = 0; i < 4; i++) {
			if (sd->RyR_Neighb[idx*4 + i] >= 0) {
				if (sd->RyR_States[sd->RyR_Neighb[idx*4 + i]]) {
					N_Open++;
				} else {
					N_Closed++;
				}
			}
		}

		double X, rate;
		double ca_ss = sd->Grid.States[INDEX_CA][sd->RyR_Ele[idx]];

		if (sd->RyR_States[idx]) {
			//Open -> Closed rate
			X = exp( 0.5*ps->Reals[I_RYR_A_STAR]*(N_Open*ps->Reals[I_RYR_EPS_OO] - N_Closed*ps->Reals[I_RYR_EPS_CC]) );
			rate = X*ps->Reals[I_RYR_K_MINUS];

		} else {
			//Closed -> Open rate
			double phi = ps->Reals[I_RYR_PHI_M]*sd->Grid.States[INDEX_CA][sd->RyR_JSR_Ele[idx]] + ps->Reals[I_RYR_PHI_B];
			X = exp( 0.5*ps->Reals[I_RYR_A_STAR]*(N_Closed*ps->Reals[I_RYR_EPS_CC] - N_Open*ps->Reals[I_RYR_EPS_OO]) );
			rate = X*phi*ps->Reals[I_RYR_K_PLUS]* pow(ca_ss,ps->Reals[I_RYR_ETA]);
		}
		RyR_Rates[idx] = rate;
		Rate_Total += rate;
	}

	//Randomly choose RyR to open
	int seed = 93843 + ps->Integers[I_RAND_SEED_0] + sd->device*ps->Integers[I_RAND_ADD_A] + sd->ensemble_index*ps->Integers[I_RAND_ADD_B];
	if (ps->Integers[I_FLAG_CLOCK_SEED]) {
		seed += (int)time(NULL);
	}
	srand(seed);
	double r = Rate_Total*((double)rand())/((double)RAND_MAX);
	int RyR_Open = 0;
	double sum = 0;
	for (int i = 0; i < sd->N_RyR; i++) {
		sum += RyR_Rates[i];
		if (sum >= r) {
			RyR_Open = i;
			break;
		}
	}
	fprintf(stdout,"Opening RyR #%d...\n",RyR_Open);
	sd->RyR_States[RyR_Open] = 1;
	delete[] RyR_Rates;
}
